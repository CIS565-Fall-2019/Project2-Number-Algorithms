#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

//void memory_debug_float(int elements, float* cuda_mem, float* cpu_mem)
//{
//	printf("elements %d\n ", elements);
//	hipMemcpy(cpu_mem, cuda_mem, elements * sizeof(float), hipMemcpyDeviceToHost);
//	checkCUDAErrorFn("debug failed!");
//	printf("=============================\n");
//	for (int i = 0; i < elements; i++)
//	{
//		printf("out[%d] %d \n", i, cpu_mem[i]);
//	}
//	printf("=============================\n");
//}