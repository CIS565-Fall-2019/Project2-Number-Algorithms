#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %d: %s: %s\n", line, msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

/**
* Constructor (empty)
*/
InputData::InputData(void) {
	data = uint8_v();
	fData = float_v();
	resultArray = float_v();
}//empty constructor

void InputData::fillArray(uint8_t* dest) {
	memcpy(dest, this->data.data(), this->numElements * sizeof(uint8_t));
}//fillArray

void InputData::fillActivationArray(float_v* dest) {

	dest->clear();

	for (int i = 0; i < RSIZE; i++) {
		if (i == this->value) {
			dest->push_back(1.0);
		}//if
		else {
			dest->push_back(0.0);
		}//else
	}//for
}//fillActivationArray

char getShade(float input, float scale = 1.0) {
	std::string shadeString = std::string(" .:-=+*#%@");
	int numLevels = shadeString.size();
	float step = 1.0 / numLevels;

	float normInput = input / scale;
	if (normInput < 0.0) normInput = 0.0;
	if (normInput >= 1.0) normInput = 0.99;

	int inputStep = (int)(normInput / step);
	return shadeString.at(inputStep);

}//getShade

void printFloatPic(float* begin, int width, int height) {
	printf("\n");
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			float rawVal = begin[i * width + j];
			printf("%c", getShade(rawVal));
		}//for
		printf("\n");
	}//for
	printf("\n");
}//printFloatPic