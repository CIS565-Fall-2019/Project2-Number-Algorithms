#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>

namespace CharacterRecognition {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	int blockSize = 128;
	dim3 threadsPerBlock(blockSize);

	__host__ __device__ unsigned int hash(unsigned int a) {
		a = (a + 0x7ed55d16) + (a << 12);
		a = (a ^ 0xc761c23c) ^ (a >> 19);
		a = (a + 0x165667b1) + (a << 5);
		a = (a + 0xd3a2646c) ^ (a << 9);
		a = (a + 0xfd7046c5) + (a << 3);
		a = (a ^ 0xb55a4f09) ^ (a >> 16);
		return a;
	}

	__global__ void kernFillRandom(int N, float *weights, float time) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= N) {
			return;
		}

		
		thrust::default_random_engine rng(hash((int)(index * time)));
		thrust::uniform_real_distribution<float> unitDistrib(-12, 12);


		weights[index] = (float)unitDistrib(rng);
	}


	void fillRandomWeights(int n, float *data, float seed) {
		float *dev_weightsArray;

		hipMalloc((void**)&dev_weightsArray, n * sizeof(float));
		checkCUDAError("hipMalloc dev_weightsArray failed!");

		int numThreads = n;
		dim3 blocksPerGrid((numThreads + blockSize - 1) / blockSize);

		kernFillRandom<<<blocksPerGrid, threadsPerBlock>>>(numThreads, dev_weightsArray, seed);
		checkCUDAError("kernFillRandom failed!");

		hipMemcpy(data, dev_weightsArray, n * sizeof(float), hipMemcpyDeviceToHost);

		hipFree(dev_weightsArray);
	}


	__global__ void kernLayer1Mult(int numHidden, float *hiddenLayers, int inputSize, const float* input, const float *weights) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= numHidden) {
			return;
		}
		float sum = 0;
		for (int i = 0; i < inputSize; ++i) {
			sum += input[i] * weights[index + numHidden * i];
		}

		hiddenLayers[index] = 1 / (1 + exp(-sum));
	}

	__global__ void kernLayer2Mult(int n, int numHiddenlayers, float *output, const float *hiddenLayers, const float *weights) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= n) {
			return;
		}
		float sum = 0;
		for (int i = 0; i < numHiddenlayers; ++i) {
			sum += hiddenLayers[i] * weights[i];
		}
		output[index] = 1 / (1 + exp(-sum));
	}

	__global__ void kernPartialErrorDeriv1(int n,
		float expectedValue, float output, int inputSize, int numHidden,
		const float *input, const float *hidden, const float *weights2, float *partials1) {
		
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= n) {
			return;
		}

		//float originalWeight = adjustedWeights[index]; // Do the memory acces first and let the following math hide latency

		int inputIndex = floorf(index / (numHidden));
		int hiddenIndex = index % numHidden;

		float inputValue = input[inputIndex];
		float hiddenValue = hidden[hiddenIndex];
		float hiddenWeight = weights2[hiddenIndex];

		float partialErrorDeriv = -inputValue * (1 / (1 + exp(-hiddenValue))) *
			(1 - (1 / (1 + exp(-hiddenValue)))) * (expectedValue - output) *
			(1 / (1 + exp(-output))) * (1 - (1 / (1 + exp(-output)))) *
			hiddenWeight;

		//float deltaWeight = (error / 10.0) * partialErrorDeriv;

		partials1[index] = partialErrorDeriv;
	}

	__global__ void kernPartialErrorDeriv2(int n,
		float expectedValue, float output,
		const float *hidden, float *partials2) {

		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= n) {
			return;
		}

		//float originalWeight = adjustedWeights[index];

		float partialErrorDeriv = (-(expectedValue - output)) * (1 / (1 + exp(-output))) * (1 - (1 / (1 + exp(-output)))) * hidden[index];

		//float deltaWeight = (error / 10.0) * partialErrorDeriv;

		partials2[index] =  partialErrorDeriv;



	}


	float mlp(int inputSize, int numHiddenLayers, float expectedValue, 
		const float *weights1, const float *weights2, 
		const float *idata, 
		float *partialDerivatives1, float *partialDerivatives2) {
		// size of input is 2 for xor and 512 by 512 for characters
		// hidden layer somewhere between 1 and size of input
		// first number of weights is size of hidden layer * size of input
		// second number of weights is size of hidden layer * size of output(1)

		int numWeights1 = inputSize * numHiddenLayers;
		int numWeights2 = numHiddenLayers;


		// Initialize buffers
		float *dev_inputData;
		float *dev_hidden;
		float *dev_weights1;
		float *dev_weights2;
		float *dev_output;

		float *dev_partials1;
		float *dev_partials2;

		float *host_output;

		float *host_hidden;

		// Malloc for buffers
		hipMalloc((void**)&dev_inputData, inputSize * sizeof(float));
		checkCUDAError("hipMalloc dev_inputData failed!");
		
		hipMalloc((void**)&dev_hidden, numHiddenLayers * sizeof(float));
		checkCUDAError("hipMalloc dev_hidden failed!");

		hipHostMalloc((void**)&host_hidden, numHiddenLayers * sizeof(float));
		checkCUDAError("hipHostMalloc host_hidden failed!");
		
		hipMalloc((void**)&dev_weights1, numWeights1 * sizeof(float));
		checkCUDAError("hipMalloc dev_weights1 failed!");
		
		hipMalloc((void**)&dev_weights2, numWeights2 * sizeof(float));
		checkCUDAError("hipMalloc dev_weights2 failed!");

		hipMalloc((void**)&dev_partials1, numWeights1 * sizeof(float));
		checkCUDAError("hipMalloc dev_partials1 failed!");

		hipMalloc((void**)&dev_partials2, numWeights2 * sizeof(float));
		checkCUDAError("hipMalloc dev_partials2 failed!");

		hipMalloc((void**)&dev_output, sizeof(float));
		checkCUDAError("hipMalloc dev_output failed!");

		hipHostMalloc((void**)&host_output, sizeof(float));
		checkCUDAError("hipHostMalloc host_output failed!");

		// Fille input and weights data
		hipMemcpy(dev_inputData, idata, inputSize * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_weights1, weights1, numWeights1 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_weights2, weights2, numWeights2 * sizeof(float), hipMemcpyHostToDevice);
		
		// Perform the multiplications for layer 1 to get the hidden layers
		int numThreads = numHiddenLayers;
		dim3 blocksPerGrid((numThreads + blockSize - 1) / blockSize);
		
		kernLayer1Mult<<<blocksPerGrid, threadsPerBlock>>>(numHiddenLayers, dev_hidden, inputSize, dev_inputData, dev_weights1);
	
		// perform the multiplications for layer 2 to get the output value
		int layer2_numThreads = 1;
		dim3 layer2_blocksPerGrid((layer2_numThreads + blockSize - 1) / blockSize);

		kernLayer2Mult<<<layer2_blocksPerGrid, threadsPerBlock>>>(1, numHiddenLayers, dev_output, dev_hidden, dev_weights2);

		// Copy the output onto the host
		hipMemcpy(host_output, dev_output, sizeof(float), hipMemcpyDeviceToHost);
		float output = host_output[0];

		// Find the error from the output
		//float error = (output - expectedValue) * (output - expectedValue);
		//std::cout << "error " << error << std::endl;

		// Adjust the weights of the layer 1 weights
		int weight1Adjust_numThreads = numWeights1;
		dim3 weight1Adjust_blocksPerGrid((weight1Adjust_numThreads + blockSize - 1) / blockSize);

		kernPartialErrorDeriv1<<<weight1Adjust_blocksPerGrid, threadsPerBlock>>>(numWeights1, expectedValue, 
			output, inputSize,
			numHiddenLayers, dev_inputData, dev_hidden, 
			dev_weights2, dev_partials1);

		// Copy the weights into the input array
		hipMemcpy(partialDerivatives1, dev_partials1, numWeights1 * sizeof(float), hipMemcpyDeviceToHost);


		// Adjust the weights of the layer 2 weights
		int weight2Adjust_numThreads = numWeights2;
		dim3 weight2Adjust_blocksPerGrid((weight2Adjust_numThreads + blockSize - 1) / blockSize);

		kernPartialErrorDeriv2<<<weight2Adjust_blocksPerGrid, threadsPerBlock>>>(numWeights2, 
			expectedValue, output, dev_hidden, dev_partials2);

		hipMemcpy(partialDerivatives2, dev_partials2, numWeights2 * sizeof(float), hipMemcpyDeviceToHost);

		//for (int i = 0; i < numWeights1; ++i) {
		//	//std::cout << "adjusted weight: " << adjustedWeights1[i] << std::endl;
		//}


		// Free buffer memory
		hipFree(dev_inputData);
		hipFree(dev_hidden);
		hipFree(dev_weights1);
		hipFree(dev_weights2);
		hipFree(dev_partials1);
		hipFree(dev_partials2);
		hipFree(dev_output);
		hipHostFree(host_output);
		hipHostFree(host_hidden);


		return output;

	}


	__global__ void kernAddDelta(int n, float accumulatedError, const float *partials,
		float *weights) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= n) {
			return;
		}

		float delta = -(accumulatedError / 5.0) * partials[index];
		weights[index] += delta;
	}

	void updateWeights(int numWeights, float accumulatedError, const float *partials, float *weights) {
		float *dev_partials;
		float *dev_weights;

		hipMalloc((void**)&dev_partials, numWeights * sizeof(float));
		checkCUDAError("hipMalloc dev_partials failed!");
		
		hipMalloc((void**)&dev_weights, numWeights * sizeof(float));
		checkCUDAError("hipMalloc dev_weights failed!");

		hipMemcpy(dev_partials, partials, numWeights * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_weights, weights, numWeights * sizeof(float), hipMemcpyHostToDevice);


		int numThreads = numWeights;
		dim3 blocksPerGrid((numThreads + blockSize - 1) / blockSize);

		kernAddDelta<<<blocksPerGrid, threadsPerBlock>>>(numWeights, accumulatedError, dev_partials, dev_weights);

		hipMemcpy(weights, dev_weights, numWeights * sizeof(float), hipMemcpyDeviceToHost);

		hipFree(dev_weights);
		hipFree(dev_partials);


	}

}
