#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define blockSize 128
#define blockWidth 16


namespace CharacterRecognition {
	using Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}

	// Initlialiations

	//layers
	double *dev_iLayer;
	double *dev_hLayer;
	double *dev_oLayer;

	double *dev_b1;
	double *dev_b2;
	double *dev_db1;
	double *dev_db2;

	double *dev_losses;
	double *dev_LossAvg;

	// gtruth and preds
	int *dev_gtruth;
	int *dev_preds;
	double * dev_preds_probab;

	//weights
	double *dev_w_kj;
	double *dev_w_ji;

	//Derivatives
	double *dev_dL_dw_ji;
	double *dev_dL_dw_kj;
	double *dev_dL_dscores;
	double *dev_dL_dscores_2;

	double *dev_hLayer_T;
	double *dev_iLayer_T;
	double *dev_w_ji_T;


	//=============================================
	// Rnadom Number Generation using cuRand on GPU
	//=============================================
	hiprandState *devState;

	__global__ void kernInitCurand(hiprandState *state, int N, unsigned long seed) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < N) {
			hiprand_init(seed, tid, 0, &state[tid]);
		}
	}

	__global__ void KernGenRand(hiprandState *state, int N, double *w) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < N) {
			w[tid] = (2.0*hiprand_uniform(&state[tid]) - 1.0); // Between -1 and 1
		}
	}

	//===================================================================
	//=====KERNEL DEFNITIONS FOR Forward and Backward====================
	//===================================================================


	void printArray(int n, int *a, bool abridged = false) {
		printf("    [ ");
		for (int i = 0; i < n; i++) {
			if (abridged && i + 2 == 15 && n > 16) {
				i = n - 2;
				printf("... ");
			}
			printf("%3d ", a[i]);
		}
		printf("]\n\n");
	}
	void printFloatArray(int n, double *a, bool abridged = false) {
		printf("    [ ");
		for (int i = 0; i < n; i++) {
			if (abridged && i + 2 == 15 && n > 16) {
				i = n - 2;
				printf("... ");
			}
			printf("%3f ", a[i]);
		}
		printf("]\n\n");
	}



	// Kernel for Gradient update on Weights
	__global__ void kernUpdateWeights(int N, double *dev_dw, double *dev_w, double LR) {

		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			dev_w[tid] = dev_w[tid] - (LR * dev_dw[tid]);
		}
	}

	// Kernel for derivative of sigmoid
	__global__ void kernGradSigmoid(int N, int H, double *dev_hLayer) {

		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N*H) {
			dev_hLayer[tid] = dev_hLayer[tid] * (1 - dev_hLayer[tid]);
		}
	}

	// Matrix Transpose
	__global__ void kernMatrixTranspose(int rows, int cols, double *matrix, double *matrix_T) {

		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int idx = blockIdx.x * blockDim.x + threadIdx.x;

		if (idx < cols && idy < rows) {
			int pos = idy * cols + idx;
			int tpos = idx * rows + idy;

			matrix_T[tpos] = matrix[pos];
		}
	}

	// Divide by N
	__global__ void kernDivNdscores(int N, int C, double *dev_dL_dscores) {

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < N*C) {
			dev_dL_dscores[tid] /= N;
		}
	}

	// Compute dscores gradient
	__global__ void kernSetdscores(int N, int C, double *dev_dL_dscores, int *dev_gtruth) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			dev_dL_dscores[tid*C + dev_gtruth[tid]] -= 1;
		}
	}

	// compute predictions
	__global__ void kernPredsN(int N, int C, double* dev_oLayer, int* dev_gtruth, int* dev_preds, double * dev_preds_probab) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			dev_preds[tid] = dev_oLayer[tid*C + dev_gtruth[tid]] > 0.5 ? dev_gtruth[tid] : (dev_gtruth[tid] == 0 ? 1 : 0);
			dev_preds_probab[tid] = dev_oLayer[tid*C + dev_gtruth[tid]];
		}
	}

	// compute loss per example
	__global__ void kernLossPerN(int N, int C, double* dev_oLayer, int* dev_gtruth, double* dev_losses) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		
		if (tid < N) {
			//printf("tid = %d \n", tid);
			//printf("tid*C = %d \n", tid*C);
			//printf("tid*C + gtruth = %d \n", tid*C + dev_gtruth[tid]);
			//printf("dev_oLayer[tid*C + dev_gtruth[tid]] = %0.3f \n", dev_oLayer[ tid*C + dev_gtruth[tid] ]);
			//printf("dev_oLayer[tid*C + dev_gtruth[tid]] = %0.3f \n", log(dev_oLayer[tid*C + dev_gtruth[tid]]));
			dev_losses[tid] = -log(dev_oLayer[tid*C + dev_gtruth[tid]]);

		}
	}

	// kernel to compute exp softmax
	__global__ void kernSoftmax(int N, int C, double* scores) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			double sums = 0.0;

			for (int i = 0; i < C; i++) {
				sums += exp(scores[tid*C + i]);
			}

			for (int i = 0; i < C; i++) {
				scores[tid*C + i] = exp(scores[tid*C + i]) / sums;
			}
		}
	}

	// kern for sigmoid // f(x) = 1/(1 + e^-x).
	__global__ void kernSigmoid(int N, double *idata) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;

		if (tid < N) {
			idata[tid] = 1.0 / (1.0 + exp(-1*idata[tid]));
		}
	}

	// kern for element wise product 
	__global__ void kernElementProduct(int N, double *matrixA, double* matrixB) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;

		if (tid < N) {
			matrixA[tid] = matrixA[tid] * matrixB[tid];
		}
	}


	// kernel to to matmul // A mxn // B nxk // C mxk
	__global__ void kernMatrixMultiply(const double *dev_A, const double *dev_B, double *dev_C, int m, int n, int k) {

		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;

		double sum = 0;
		if (col < k && row < m)
		{
			for (int i = 0; i < n; i++)
				sum += dev_A[row * n + i] * dev_B[i * k + col];
			dev_C[row * k + col] = sum;
		}
	}

	// Dumb reduction
	__global__ void kernReduction(int N, double *dev_losses, double *dev_LossAvg) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		double sum = 0.0;
		if (tid == 0) {
			for (int i = 0; i < N; i++) {
				sum += dev_losses[i];
			}
			dev_LossAvg[0] = sum / N;
		}

	}

	// Ele wise addition A = A+B
	__global__ void kernAddition(int N, double *dev_A, double *dev_B) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;

		if (tid < N) {
			dev_A[tid] += dev_B[tid];
		}

	}

	void trainMLP(int N, int D, int H, int C, double *idata, int *preds, int *gtruth, int epochs, double *lossAvgPerEpoch, const double LR, unsigned long seed) {

		timer().startGpuTimer();

		// N = number of examples
		// D = dim of each example 
		// C = number of classes

		// NETWORK DEFITION_____________
		// Compute f1		= W1*X1
		// Compute X2		= Sig(f1)
		// Compute Scroes S = W2*X2
		// Compute Probab P = Softmax(S)
		// Compute Loss   L = CEntropy(P)

		//================================================================
		//======================INITIALIZATIONS===========================
		//================================================================

		// Allocate input layer
		hipMalloc((void**)&dev_iLayer, N*D * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_iLayer failed!");

		hipMemcpy(dev_iLayer, idata, N*D * sizeof(double), hipMemcpyHostToDevice);
		checkCUDAErrorFn("hipMemcpyToSymbol from idata to dev_iLayer failed!");


		// Allocate hidden layer
		hipMalloc((void**)&dev_hLayer, N*H* sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer failed!");


		// Allocate output layer
		hipMalloc((void**)&dev_oLayer, N*C* sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_oLayer failed!");


		// Allocate losses holder
		hipMalloc((void**)&dev_losses, N * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_losses failed!");

		hipMalloc((void**)&dev_LossAvg, 1*sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_LossAvg failed!");


		// Allocate gtruth and preds
		hipMalloc((void**)&dev_gtruth, N * sizeof(int));
		checkCUDAErrorFn("hipMalloc dev_gtruth failed!");

		hipMemcpy(dev_gtruth, gtruth, N * sizeof(int), hipMemcpyHostToDevice);
		checkCUDAErrorFn("hipMemcpyToSymbol from gtruth to dev_gtruth failed!");

		hipMalloc((void**)&dev_preds, N * sizeof(int));
		checkCUDAErrorFn("hipMalloc dev_preds failed!");

		hipMalloc((void**)&dev_preds_probab, N * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_preds_probab failed!");

		// Allocate Weights
		hipMalloc((void**)&dev_w_kj, D*H * sizeof(double)); //w1
		checkCUDAErrorFn("hipMalloc dev_w_kj failed!");

		hipMalloc((void**)&dev_w_ji, C*H * sizeof(double)); //w2
		checkCUDAErrorFn("hipMalloc dev_w_ji failed!");


		// Allocate Derivatives
		hipMalloc((void**)&dev_dL_dw_kj, D*H * sizeof(double)); //dw1
		checkCUDAErrorFn("hipMalloc dev_w_kj failed!");

		hipMalloc((void**)&dev_dL_dw_ji, C*H * sizeof(double)); //dw2
		checkCUDAErrorFn("hipMalloc dev_w_ji failed!");

		hipMalloc((void**)&dev_dL_dscores, N*C * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_dL_dscores failed!");

		hipMalloc((void**)&dev_dL_dscores_2, N*C * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_dL_dscores_2 failed!");


		// Allocate transposes
		hipMalloc((void**)&dev_hLayer_T, N*H * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");

		hipMalloc((void**)&dev_iLayer_T, N*D * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");

		hipMalloc((void**)&dev_w_ji_T, C*H * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_w_ji_T failed!");

		/*
		//Allocate biases
		hipMalloc((void**)&dev_b1, N*H * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");

		hipMalloc((void**)&dev_b2, N*C * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");

		hipMalloc((void**)&dev_db1, N*H * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");

		hipMalloc((void**)&dev_db2, N*C * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");
		*/

		//==============================
		// Initialise Weights and Biases
		//==============================
		hipMalloc((void**)&devState, H*D * sizeof(hiprandState));

		kernInitCurand << <((D*H + blockSize - 1) / blockSize), blockSize >> > (devState, D*H, seed);
		checkCUDAErrorFn("KernInitCurand failed!");
		KernGenRand << <((D*H + blockSize - 1) / blockSize), blockSize >> > (devState, D*H, dev_w_kj);//w1
		checkCUDAErrorFn("KernGenRand dev_w_kj failed!");

		kernInitCurand << <((H*C + blockSize - 1) / blockSize), blockSize >> > (devState, H*C, seed); 
		checkCUDAErrorFn("KernInitCurand failed!");
		KernGenRand << <((H*C + blockSize - 1) / blockSize), blockSize >> > (devState, H*C, dev_w_ji);//w2
		checkCUDAErrorFn("KernGenRand dev_w_kj failed!");

		/*
		kernInitCurand << <((N*C + blockSize - 1) / blockSize), blockSize >> > (devState, N*C, 9); //9
		checkCUDAErrorFn("KernInitCurand failed!");
		KernGenRand << <((N*C + blockSize - 1) / blockSize), blockSize >> > (devState, N*C, dev_b2);
		checkCUDAErrorFn("KernGenRand dev_w_kj failed!");

		kernInitCurand << <((N*H + blockSize - 1) / blockSize), blockSize >> > (devState, N*H, 9999); //9999
		checkCUDAErrorFn("KernInitCurand failed!");
		KernGenRand << <((N*H + blockSize - 1) / blockSize), blockSize >> > (devState, N*H, dev_b1);
		checkCUDAErrorFn("KernGenRand dev_w_kj failed!");
		*/

		/*double *rand = new double[D*C];
		hipMemcpy(rand, dev_w_kj, D*C* sizeof(double), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpyFromSymbol from dev_w_kj to rand failed!");
		printf("Post random inits dev_w_kj - \n");
		printFloatArray(D*C, rand, true);*/

		/*double *rand2 = new double[C*C];
		hipMemcpy(rand2, dev_w_ji, C*C * sizeof(double), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpyFromSymbol from dev_w_kj to rand failed!");
		printf("Post random inits dev_w_ji - \n");
		printFloatArray(C*C, rand2, true);*/

		//================================================================
		//======================TRAINING LOOP=============================
		//================================================================
		double *tmp = new double[N*D];
		double *tmp2 = new double[N*D];
		double *lossesN = new double[N];
		int *tmpint = new int[N];

		printf("Input DATA\n");
		printFloatArray(N*D, idata, true);
		dim3 dimBlock(blockWidth, blockWidth);
		dim3 dimGrid;

		for (int i = 0; i < epochs; i++) {

			//================================================================
			//========================= FORWARD ==============================

			// STEP 1
			// f1 = W1*X1 (Matrix Mul)
			//=================================
			// dev_hLayer = dev_iLayer*dev_w_kj 
			//   NxH      =    NxD         DxH 


			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_iLayer, dev_w_kj, dev_hLayer, N, D, H);

			//kernAddition << < ((N*H + blockSize - 1) / blockSize), blockSize >> > (N*H, dev_hLayer, dev_b1);

			// Copy back to cpu
			//double *tmp = new double[N*H];
			//hipMemcpy(tmp, dev_hLayer, N*H* sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol from dev_arrayA to odata failed!");
			//printf("Post matmul [f1 = dev_iLayer*dev_w_kj]\n");
			//printFloatArray(N*H, tmp, true);

			// STEP 2
			// X2         = Sigmoid(f1) 
			//================================
			// dev_hLayer = sigmoid(dev_hLayer)
			//   NxH     =    NxH 
			kernSigmoid << <((N*H + blockSize - 1) / blockSize), blockSize >> > (N*H, dev_hLayer);


			// Copy back to cpu
			//hipMemcpy(tmp, dev_hLayer, N*H*sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol from dev_arrayA to odata failed!");
			//printf("Post sigmoid [X2 = Sigmoid(f1) ]\n");
			//printFloatArray(N*H, tmp, true);

			// STEP 3
			// Scores S = W2*X2 (Matrix Mul)
			//================================
			// dev_oLayer = dev_hLayer*dev_w_ji 
			//   NxC      =    NxH         HxC
			dimGrid.x = (C + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_hLayer, dev_w_ji, dev_oLayer, N, H, C);
			//kernAddition << < ((N*C + blockSize - 1) / blockSize), blockSize >> > (N*C, dev_oLayer, dev_b2);
			checkCUDAErrorFn("kernMatrixMultiply failed!");

			// Copy back to cpu
			//hipMemcpy(tmp, dev_oLayer, N*C*sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol from dev_arrayA to odata failed!");
			//printf("Post S=W2*x2\n");
			//printFloatArray(N*C, tmp, true);

			// STEP 4
			// P = Softmax(S) 
			//===============
			// dev_smaxDen = Sum_Over_classses(dev_olayer)
			// dev_olayer = dev_olayer/Sum_Over_classses
			//   NxC      =    NxC         1
			kernSoftmax << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_oLayer);
			checkCUDAErrorFn("kernSoftmax failed!");

			// Copy back to cpu
			hipMemcpy(tmp, dev_oLayer, N*C * sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyFromSymbol from dev_oLayer to tmp failed!");
			printf("Post Softmax  [dev_olayer = exp(dev_olayer)/Sum_Over_classses]\n");
			printFloatArray(N*C, tmp, true);

			// print check
			//hipMemcpy(tmpint, dev_gtruth, N * sizeof(int), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol from dev_gtruth to tmpint failed!");
			//printf("Print gtruth \n");
			//printArray(N, tmpint, true);

			// STEP 5
			// Compute Losses | Cross Entropy Loss
			//==================================
			// Compute Loss   L = CEntropy(P)
			kernLossPerN << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_oLayer, dev_gtruth, dev_losses);
			checkCUDAErrorFn("kernLossPerN  failed!");

			// Copy back to cpu
			hipMemcpy(lossesN, dev_losses, N * sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyFromSymbol from dev_losses to lossesN failed!");
			printf("Post dev_losses [Loss = CEntropy(P)]\n");
			printFloatArray(N, lossesN, true);


			// Predictions
			kernPredsN << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_oLayer, dev_gtruth, dev_preds, dev_preds_probab);
			hipMemcpy(preds, dev_preds, N * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyDeviceToHost from dev_preds to preds failed!");
			hipMemcpy(tmp2, dev_preds_probab, N * sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyDeviceToHost from dev_preds_probab to tmp failed!");

			printf("Predictions\n");
			printArray(N, preds, true);
			printFloatArray(N, tmp2, true);


			// STEP 5.2
			// Compute Avg of Losses
			//==================================
			// Dumb Reduction

			kernReduction << <((N + blockSize - 1) / blockSize), blockSize >> > (N, dev_losses, dev_LossAvg);
			// Copy back to cpu
			hipMemcpy(lossAvgPerEpoch + i, dev_LossAvg, sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyFromSymbol from dev_LossAvg to tmp failed!");

			printf("Epoch: %d | LossAvg %3f \n", i, lossAvgPerEpoch[i]);

			
			//=================================================================
			//========================= BACKPROP ==============================

			// STEP 1 : Gradient wrt w_ji
			// dW_ji = Probs_k - [1](gth == k) dev_dL_dscores; 
			hipMemcpy(dev_dL_dscores, dev_oLayer, N*C * sizeof(double), hipMemcpyDeviceToDevice);
			checkCUDAErrorFn("hipMemcpyFromSymbol from probabs to dev_dL_dscores failed!");


			kernSetdscores << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_dL_dscores, dev_gtruth);
			checkCUDAErrorFn("kernSetdscores failed!");

			//hipMemcpy(tmp, dev_dL_dscores, N*C * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol from dev_oLayer to dev_dL_dscores failed!");
			//printf("dev_dL_dscores \n");
			//printFloatArray(N*C, tmp, true);

			// Copy back to cpu
			//hipMemcpy(tmp, dev_dL_dscores, N*C * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol [kernSetdscores] from dev_dL_dscores to tmp failed!");
			//printf("Post setting loss at positions dev_dL_dscores \n");
			//printFloatArray(N*C, tmp, true);

			kernDivNdscores << <((N*C + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_dL_dscores);
			checkCUDAErrorFn("kernDivNdscores failed!");


			//hipMemcpy(tmp, dev_dL_dscores, N*C * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol from dev_oLayer to dev_dL_dscores failed!");
			//printf("dev_dL_dscores /N\n");
			//printFloatArray(N*C, tmp, true);


			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixTranspose << <dimGrid, dimBlock >> > (N, H, dev_hLayer, dev_hLayer_T);

			/*
			hipMemcpy(tmp, dev_hLayer, N*H * sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyFromSymbol dev_hLayer to tmp failed!");
			printf("dev_hLayer \n");
			printFloatArray(N*H, tmp, true);

			hipMemcpy(tmp, dev_hLayer_T, N*H* sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyFromSymbol dev_hLayer_T to tmp failed!");
			printf("dev_hLayer_T \n");
			printFloatArray(N*H, tmp, true);*/


			dimGrid.x = (C + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (H + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_hLayer_T, dev_dL_dscores, dev_dL_dw_ji, H, N, C);
			checkCUDAErrorFn("kernMatrixMultiply for dev_dL_dw_ji failed!");


			//===========================
			// STEP 2 : Gradient wrt w_kj
			//===========================

			// Transpose Wji (W2)
			dimGrid.x = (C + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (H + dimBlock.y - 1) / dimBlock.y;
			kernMatrixTranspose << <dimGrid, dimBlock >> > (H, C, dev_w_ji, dev_w_ji_T);

			// Transpose Input Data
			dimGrid.x = (D + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixTranspose << <dimGrid, dimBlock >> > (N, D, dev_iLayer, dev_iLayer_T);

			// Mul dev_dL_dscores * dev_w_kj_T == dev_dL_dscores_2
			//             NxC          CxH             NxH
			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_dL_dscores, dev_w_ji_T, dev_dL_dscores_2, N, C, H);
			checkCUDAErrorFn("kernMatrixMultiply for dev_dL_dscores_2 failed!");

			// compute sig gradient on dev_hlayer N*H [IN PLACE]
			kernGradSigmoid << <((N*H + blockSize - 1) / blockSize), blockSize >> > (N, H, dev_hLayer);
			checkCUDAErrorFn("kernGradSigmoid failed!");


			//Element wise mul dev_dL_dscores_2 [INPLACE] = dev_dL_dscores_2 . dev_hlayer[sig gradient] 
			kernElementProduct << <((N*H + blockSize - 1) / blockSize), blockSize >> > (N*H, dev_dL_dscores_2, dev_hLayer);
			checkCUDAErrorFn("kernElementProduct failed!");

			// matrix Mul final with Xi_T
			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (D + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_iLayer_T, dev_dL_dscores_2, dev_dL_dw_kj, D, N, H);
			checkCUDAErrorFn("kernMatrixMultiply for dev_dL_dw_kj failed!");


			//=================================================================
			//========================= Update Weights=========================

			// Update weights kj W1
			kernUpdateWeights << <((D*H + blockSize - 1) / blockSize), blockSize >> > (D*H, dev_dL_dw_kj, dev_w_kj, LR);
			checkCUDAErrorFn("kernUpdateWeights dev_w_kj failed!");

			// InitUpdate weights ji W2
			kernUpdateWeights << <((H*C + blockSize - 1) / blockSize), blockSize >> > (H*C, dev_dL_dw_ji, dev_w_ji, LR);
			checkCUDAErrorFn("kernUpdateWeights dev_w_ji failed!");

			/*
			// Update biases1
			kernUpdateWeights << <((N*H + blockSize - 1) / blockSize), blockSize >> > (N*H, dev_db1, dev_dL_dscores_2, LR);
			checkCUDAErrorFn("kernUpdateWeights dev_w_kj failed!");

			// InitUpdate biases2
			kernUpdateWeights << <((N*C + blockSize - 1) / blockSize), blockSize >> > (N*C, dev_db2, dev_dL_dscores, LR);
			checkCUDAErrorFn("kernUpdateWeights dev_w_ji failed!");
			*/

			// COntinue to next epoch 
			//hipMemcpy(tmp2, dev_w_kj, D*H * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("dev_w_kj memcopy failed!");
			//printf("w_kj \n");
			//printFloatArray(D*H, tmp2, true);
			//hipMemcpy(tmp2, dev_dL_dw_kj, D*H * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("dev_dL_dw_kj memcopy failed!");
			//printf("Dw_kj \n");
			//printFloatArray(D*H, tmp2, true);

			//hipMemcpy(tmp2, dev_w_ji, H*C * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("dev_w_ji memcopy failed!");
			//printf("w_ji \n");
			//printFloatArray(H*C, tmp2, true);
			//hipMemcpy(tmp2, dev_dL_dw_ji, H*C * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("dev_dL_dw_ji memcopy failed!");
			//printf("Dw_ji \n");
			//printFloatArray(H*C, tmp2, true);


			printf("\n-----------------------------------------------------\n\n");
		}


		printf("Finished training.\n");
		printf("losses:\n");
		printFloatArray(epochs, lossAvgPerEpoch, true);

		//====================
		// CleanUp
		//====================
		hipFree(dev_iLayer);
		hipFree(dev_hLayer);
		hipFree(dev_oLayer);

		hipFree(dev_losses);

		hipFree(dev_gtruth);
		hipFree(dev_preds);
		hipFree(dev_preds_probab);


		hipFree(dev_w_kj);
		hipFree(dev_w_ji);

		hipFree(dev_b1);
		hipFree(dev_b2);
		hipFree(dev_db1);
		hipFree(dev_db2);

		hipFree(dev_dL_dw_ji);
		hipFree(dev_dL_dw_kj);

		hipFree(dev_dL_dscores);
		hipFree(dev_dL_dscores_2);

		hipFree(dev_hLayer_T);
		hipFree(dev_iLayer_T);

		delete(tmp);
		delete(tmp2);

		timer().endGpuTimer();
	}
}
