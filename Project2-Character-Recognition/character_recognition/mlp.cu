#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include "common.h"
#include "mlp.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/fill.h>

#define blockSize 128

int INPUT_LAYER_SIZE;
int HIDDEN_LAYER_SIZE;
int OUTPUT_LAYER_SIZE;
float *weights_IH, *weights_HO, *g_weights_IH, *g_weights_HO, *hidden, *h_sigmoid, *output, *o_softmax;
hipblasHandle_t cublas_handle;
hiprandGenerator_t prng;

void print_matrix(const float *devA, int nr_rows_A, int nr_cols_A) {
	float *A = new float[nr_rows_A*nr_cols_A];
	hipMemcpy(A, devA, nr_rows_A*nr_cols_A * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < nr_rows_A; ++i) {
		for (int j = 0; j < nr_cols_A; ++j) {
			printf("%f \t", A[j * nr_rows_A + i]);
		}
		printf("\n");
	}
	printf("\n");
}

namespace StreamCompaction {
	__global__ void kernelUpSweepStepEfficient(int n, int d, float* cdata) {
		int k = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (k >= n)
			return;
		int prev_step_size = 1 << d;
		int cur_step_size = 2 * prev_step_size;
		int new_offset = k * cur_step_size;
		cdata[new_offset + cur_step_size - 1] += cdata[new_offset + prev_step_size - 1];
	}
	/**
	 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
	 */
	void sumArray(int n, float* sum, const float *idata) {
		// Memory Allocation and Copying
		int power_size = pow(2, ilog2ceil(n));
		float *sumArray;
		hipMalloc((void**)&sumArray, power_size * sizeof(float));
		checkCUDAErrorFn("hipMalloc sumArray failed!");
		hipMemset(sumArray, 0, power_size * sizeof(float));
		hipMemcpy(sumArray, idata, n * sizeof(float), hipMemcpyDeviceToDevice);

		int numThreads;
		//Up Sweep
		for (int d = 0; d <= ilog2ceil(power_size) - 1; d++) {
			numThreads = pow(2, (ilog2ceil(power_size) - 1 - d));
			dim3 fullBlocks((numThreads + blockSize - 1) / blockSize);
			kernelUpSweepStepEfficient << <fullBlocks, blockSize >> > (numThreads, d, sumArray);
		}
		// Copy Back and Free Memory
		hipMemcpy(sum, sumArray + power_size - 1, sizeof(float), hipMemcpyDeviceToDevice);
		hipFree(sumArray);
	}
}

namespace CharacterRecognition {

    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }
        
	// Reference: https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-hiprand-thrust/
	// Matrix Multiplication
	// nr_rows_A, nr_cols_A, nr_cols_B
	void gpu_blas_mmul(const float *A, const float *B, float *C, const int nr_rows_A, const int nr_cols_A, const int nr_cols_B) {
		int lda = nr_rows_A, ldb = nr_cols_A, ldc = nr_rows_A;
		const float alf = 1;
		const float bet = 0;
		const float *alpha = &alf;
		const float *beta = &bet;
	    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_B, nr_cols_A, alpha, A, lda, B, ldb, beta, C, ldc);
	}

	/* Forward Pass for one instance
	   1. Multiply input with input and hidden layer weights
	   2. Apply Sigmoid 
	   3. Multiply hidden layer activation with hidden and output layer weights
	   4. Apply Softmax and calculate ouput
	*/
	// TODO: Can Incorporate Bias
	void forwardPass(float* idata) {
		// Matrix Multiply Input Layer and Weights 1
		gpu_blas_mmul(idata, weights_IH, hidden, 1, INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZE);
		//printf("Hidden After: ");
		//print_matrix(hidden, 1, HIDDEN_LAYER_SIZE);
		
		// Apply Sigmoid
		dim3 hiddenLayerBlocks((HIDDEN_LAYER_SIZE + blockSize - 1) / blockSize);
		Functions::reluActivation<<<hiddenLayerBlocks, blockSize>>>(hidden, h_sigmoid, 1, HIDDEN_LAYER_SIZE);
		//printf("Hidden Sigmoid After: ");
		//print_matrix(h_sigmoid, 1, HIDDEN_LAYER_SIZE);
		
		// Matrix Multiply Hidden layer and Weights 2
		gpu_blas_mmul(h_sigmoid, weights_HO, output, 1, HIDDEN_LAYER_SIZE, OUTPUT_LAYER_SIZE);
		//printf("Output After: ");
		//print_matrix(output, 1, OUTPUT_LAYER_SIZE);
		
		// Apply Softmax
		dim3 outputLayerBlocks((OUTPUT_LAYER_SIZE + blockSize - 1) / blockSize);
		Functions::ExponentialActivation <<<outputLayerBlocks, blockSize >>> (output, o_softmax, 1, OUTPUT_LAYER_SIZE);
		float *sum;
		hipMalloc((void**)&sum, sizeof(float));
		StreamCompaction::sumArray(OUTPUT_LAYER_SIZE, sum, o_softmax);
		Functions::Divide << <outputLayerBlocks, blockSize >> > (o_softmax, sum, 1, OUTPUT_LAYER_SIZE);
		printf("Output Probabilities: ");
		print_matrix(o_softmax, 1, OUTPUT_LAYER_SIZE);
	}

	void backwardPropagation(float* dev_input, float* dev_output, float* learning_rate) {
		// Memory Allocation
		float *temp_hidden, *temp_output;
		hipMalloc((void**)&temp_output, OUTPUT_LAYER_SIZE * sizeof(float));
		hipMalloc((void**)&temp_hidden, HIDDEN_LAYER_SIZE * sizeof(float));

		// Gradient of Loss w.r.t Weight2
		dim3 outputLayerBlocks((OUTPUT_LAYER_SIZE + blockSize - 1) / blockSize);
		Functions::ElementwiseSubtraction << <outputLayerBlocks, blockSize >> > (o_softmax, dev_output, temp_output, 1, OUTPUT_LAYER_SIZE);
		gpu_blas_mmul(h_sigmoid, temp_output, g_weights_HO, HIDDEN_LAYER_SIZE, 1, OUTPUT_LAYER_SIZE);

		// Gradient of Loss w.r.t Weight1
		gpu_blas_mmul(weights_HO, temp_output, temp_hidden, HIDDEN_LAYER_SIZE, OUTPUT_LAYER_SIZE, 1);
		dim3 hiddenLayerBlocks((HIDDEN_LAYER_SIZE + blockSize - 1) / blockSize);
		Functions::KernelElementwiseMultiplyRelu << <outputLayerBlocks, blockSize >> > (temp_hidden, h_sigmoid, 1, HIDDEN_LAYER_SIZE);
		gpu_blas_mmul(dev_input, temp_hidden, g_weights_IH, INPUT_LAYER_SIZE, 1, HIDDEN_LAYER_SIZE);

		// Gradient Updates
		dim3 IHBlocks(((INPUT_LAYER_SIZE * HIDDEN_LAYER_SIZE) + blockSize - 1) / blockSize);
		Functions::Multiply << <IHBlocks, blockSize >> > (g_weights_IH, learning_rate, INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZE);
		Functions::ElementwiseSubtraction << <IHBlocks, blockSize >> > (weights_IH, g_weights_IH, weights_IH, INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZE);
		dim3 HOBlocks(((HIDDEN_LAYER_SIZE * OUTPUT_LAYER_SIZE) + blockSize - 1) / blockSize);
		Functions::Multiply << <HOBlocks, blockSize >> > (g_weights_HO, learning_rate, HIDDEN_LAYER_SIZE, OUTPUT_LAYER_SIZE);
		Functions::ElementwiseSubtraction << <HOBlocks, blockSize >> > (weights_HO, g_weights_HO, weights_HO, HIDDEN_LAYER_SIZE, OUTPUT_LAYER_SIZE);

		// Free Memory
		hipFree(temp_hidden);
		hipFree(temp_output);
	}

	float calculateLoss(int* label, int* prediction) {
		return -1;
	}

	void train(float* idata, float* ilabel, int num_instances, int epochs, float learning_rate) {
		// Create Device Buffers for Input and Output
		float *dev_input, *dev_output, *dev_lr;
		hipMalloc((void**)&dev_input, num_instances * INPUT_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc dev_input failed!");
		hipMemcpy(dev_input, idata, num_instances * INPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_output, num_instances * OUTPUT_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc dev_output failed!");
		hipMemcpy(dev_output, ilabel, num_instances * OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_lr, sizeof(float));
		thrust::device_ptr<float> dev_ptr(dev_lr);
		thrust::fill(dev_ptr, dev_ptr + 1, learning_rate);

		// Train
		for (int e = 0; e < epochs; e++) {
			for (int i = 0; i < num_instances; i++) {
				printf("Input: ");
				print_matrix(dev_input + (i * INPUT_LAYER_SIZE), 1, INPUT_LAYER_SIZE);
				// Forward Pass
				forwardPass(dev_input + (i * INPUT_LAYER_SIZE));
				// Back Propagation
				backwardPropagation(dev_input + (i * INPUT_LAYER_SIZE), dev_output + (i * OUTPUT_LAYER_SIZE), dev_lr);
			}
		}
	}

	void test(int* idata, int* ilabel, int* olabel) {

	}

	void init(int input_size, int hidden_size, int output_size) {
		printf("Init\n");
		// Initialize Layer Sizes
		INPUT_LAYER_SIZE = input_size;
		HIDDEN_LAYER_SIZE = hidden_size;
		OUTPUT_LAYER_SIZE = output_size;

		// Memory Allocation for Weight Matrices, Gradient Matrics and Hidden Layers
		hipMalloc((void**)&weights_IH, INPUT_LAYER_SIZE * HIDDEN_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc weights_IH failed!");
		hipMalloc((void**)&weights_HO, HIDDEN_LAYER_SIZE * OUTPUT_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc weights_HO failed!");
		hipMalloc((void**)&g_weights_IH, INPUT_LAYER_SIZE * HIDDEN_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc g_weights_IH failed!");
		hipMalloc((void**)&g_weights_HO, HIDDEN_LAYER_SIZE * OUTPUT_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc g_weights_HO failed!");
		hipMalloc((void**)&hidden, HIDDEN_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc hidden failed!");
		hipMalloc((void**)&h_sigmoid, HIDDEN_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc h_sigmoid failed!");
		hipMalloc((void**)&output, OUTPUT_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc output failed!");
		hipMalloc((void**)&o_softmax, OUTPUT_LAYER_SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc o_softmax failed!");

		// Create a handle for CUBLAS
		hipblasCreate(&cublas_handle);
		// Curand Random Number Generator
		hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
		// Seed for Random Number Generator
		hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	    // Initialize weight matrices with random numbers
		hiprandGenerateUniform(prng, weights_IH, INPUT_LAYER_SIZE * HIDDEN_LAYER_SIZE);
		hiprandGenerateUniform(prng, weights_HO, HIDDEN_LAYER_SIZE * OUTPUT_LAYER_SIZE);

		// Debug/Print
		print_matrix(weights_IH, INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZE);
		print_matrix(weights_HO, HIDDEN_LAYER_SIZE, OUTPUT_LAYER_SIZE);
	}

	void free() {
		hipFree(weights_IH);
		hipFree(weights_HO);
		hipblasDestroy(cublas_handle);
	}
}
