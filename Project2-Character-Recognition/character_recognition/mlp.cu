#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"
#include <thrust/random.h>

#define blockSize 512
namespace CharacterRecognition {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	void printFloatArray(float *x, int n) {
		printf("    [ ");
		for (int i = 0; i < n; i++) {
			printf("%f ", x[i]);
		}
		printf("]\n");
	}
	__host__ __device__ unsigned int hash(unsigned int a) {
	  a = (a + 0x7ed55d16) + (a << 12);
	  a = (a ^ 0xc761c23c) ^ (a >> 19);
	  a = (a + 0x165667b1) + (a << 5);
	  a = (a + 0xd3a2646c) ^ (a << 9);
	  a = (a + 0xfd7046c5) + (a << 3);
	  a = (a ^ 0xb55a4f09) ^ (a >> 16);
	  return a;
	}
	__global__ void kernInitWeightsBias(float *W, float *b, int inputDim, int outputDim){
		//Random Weight Initialization & Zero Bias Initialization
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= inputDim * outputDim) {
			return;
		}
		thrust::default_random_engine rng(hash((int)(index * inputDim * outputDim)));
		thrust::uniform_real_distribution<float> dist(0.0, 1.0);
		W[index] = dist(rng);
		//W[index] = 0.1 * index;
		int y = index / outputDim;
		b[y] = 0;
	}

	__global__ void kernAffineForward(float *W, float *b, float *in, float *out, int inputDim, int outputDim, int numSamples, bool sigmoid) {
		/*
		W: Shape inputDim x outputDim
		b: Shape outputDim
		in: Shape numSamples x inputDim
		out: Shape numSamples x outputDim
		*/
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int row = index / outputDim;
		int col = index % outputDim;
		float val = 0;
		if (row < numSamples && col < outputDim) {
			for (int i = 0; i < inputDim; i++) {
				val += in[row * inputDim + i] * W[i * outputDim + col];
			}
			val += b[row];
			out[row * outputDim + col] = sigmoid ? 1/(1+__expf(-val)) : val;
		}
	}

	__device__ float applySigmoid(float x) {
		return 1 / (1 + __expf(-x));
	}

	__device__ float dSigmoid(float x) {
		return x * (1 - x);
	}

	__global__ void kern_dSigmoid(float *dout, float *doutLinear, int numSamples, int outputDim) {
		//Apply softmax across entire dout matrix (dout is outputDim x 
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= numSamples * outputDim) {
			return;
		}
		float doutidx = dout[index];
		doutLinear[index] = doutidx * (1 - doutidx);
	}

	__global__ void kern_dIn(float *doutLinear, float *W, float *din, int inputDim, int outputDim, int numSamples) {
		/* Effectively calculates matmul(doutLinear, W.T)
		doutLinear: outputDim x numSamples - each element is dL/dY where Y = XW + b
		W: inputDim x outputDim
		din: inputDim x numSamples - each element is dL/din_(i,j)
		*/
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int row = index / inputDim;
		int col = index % inputDim;
		float val = 0;
		if (row < numSamples && col < inputDim) {
			for (int i = 0; i < outputDim; i++) {
				val += doutLinear[row * outputDim + i] * W[col * outputDim + i];
			}
			din[row * inputDim + col] = val;
		}
	}

	__global__ void kern_dW(float *W, float *b, float *doutLinear, float *in, int inputDim, int outputDim, int numSamples, float lr) {
		/* Effectively calculates matmul(input.T, doutLinear) and applies an update
		W: inputDim x outputDim (We do gradient descent here)
		b: outputDim (we do gradient decent here too)
		doutLinear: outputDim x numSamples - each element is dL/dY where Y = XW + b
		in: inputDim x numSamples
		lr: learning rate
		*/
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int row = index / outputDim;
		int col = index % outputDim;
		float val = 0;
		float dbval = 0;
		float currW = W[row * outputDim + col];
		float currb = b[col];
		float doutLinearIdx = 0;
		if (row < inputDim && col < outputDim) {
			for (int i = 0; i < numSamples; i++) {
				doutLinearIdx = doutLinear[i * outputDim + col];
				val += in[i * inputDim + row] * doutLinearIdx;
				dbval += doutLinearIdx;
			}
			W[row * outputDim + col] = currW - lr * (val);
			b[col] = currb - lr * (dbval);
		}
	}

	__global__ void kernStableSoftmax(float *pred, float *pred2, float *target, int *sums, int numSamples, int outputDim) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int row = index / outputDim;
		float rowval = 0.0;
		if (index < numSamples * outputDim) {
			for (int i = 0; i < outputDim; i++) {
				rowval += pred2[row * outputDim + i];
			}
			sums[row] = rowval;
			pred[index] = expf(pred2[index]);
			pred[index] = pred2[index] / rowval;
		}
	}

	__global__ void kernSums(float *pred, int *sums, int numSamples, int outputDim) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int row = index / outputDim;
		float rowval = 0.0;
		if (index < numSamples * outputDim) {
			for (int i = 0; i < outputDim; i++) {
				rowval += pred[row * outputDim + i];
			}
			sums[row] = rowval;
		}
	}

	//AffineLayer 
	AffineLayer::AffineLayer(int idim, int odim, int ns): numSamples(ns), inputDim(idim), outputDim(odim), sigmoid(true), eval(false), doneFwd(false){
		//Malloc Weights, Biases, in and out
		hipMalloc((void**)&W, idim * odim * sizeof(float));
		checkCUDAError("cuda Malloc W failed");
		hipMalloc((void**)&b, odim * sizeof(float));
		checkCUDAError("cuda Malloc b failed");
		hipMalloc((void**)&dev_in, inputDim * numSamples * sizeof(float));
		checkCUDAError("cuda Malloc dev_in in failed");

		//Call Initializer Kernels
		dim3 fullBlocksPerGrid((inputDim * outputDim + blockSize - 1) / blockSize);
		kernInitWeightsBias<<<fullBlocksPerGrid, blockSize>>>(W, b, inputDim, outputDim);
	}

	void AffineLayer::setSigmoid(bool state) {
		sigmoid = state;
	}
	void AffineLayer::setEval(bool state) {
		eval = state;
	}

	float* AffineLayer::forward(float *in, int ns) {
		/*Uses W & b to perform forward pass on an Affine Layer 
		Assumes dev_input is set (on GPU), numSamples is set and eval is set
		*/
		float *dev_out;
		hipMalloc((void**)&dev_out, outputDim * numSamples * sizeof(float));
		checkCUDAError("cuda Malloc dev_out in failed");

		//Memcpy the *in information into dev_in
		hipMemcpy(dev_in, in, inputDim * numSamples * sizeof(float), hipMemcpyHostToDevice);

		//Call Affine Forward Kernel 
		int numBlocks = (numSamples * outputDim + blockSize - 1) / blockSize;
		kernAffineForward<<<numBlocks, blockSize>>>(W, b, dev_in, dev_out, inputDim, outputDim, numSamples, sigmoid);

		//Memcpy out the *out and *in information from dev_out
		float *out = new float[outputDim * numSamples];
		hipMemcpy(out, dev_out, outputDim * numSamples * sizeof(float), hipMemcpyDeviceToHost);

		//free (dont free dev_in because you'll need it for backprop)
		hipFree(dev_out);
		return out;
	}

	float* AffineLayer::backward(float *dout, float lr){
		/* Does backprop and one gradient update for W & b & returns din
		dout: upstream gradient coming in 
		lr: learning rate
		Returns 
		*/
		//Malloc the input matrix and an output matrix 
		float *dev_dout, *dev_din, *dev_doutLinear;
		hipMalloc((void**)&dev_dout, outputDim * numSamples * sizeof(float));
		checkCUDAError("cuda Malloc dev_dout in failed");
		hipMalloc((void**)&dev_din, inputDim * numSamples * sizeof(float));
		checkCUDAError("cuda Malloc dev_din in failed");

		//Memcpy the *dout information into dev_dout
		hipMemcpy(dev_dout, dout, outputDim * numSamples * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("cuda Memcpy dout in failed");

		//Make 3 diff grid layouts
		dim3 weightBiasGrid((inputDim * outputDim + blockSize - 1) / blockSize);
		dim3 outputGrid = (numSamples * outputDim + blockSize - 1) / blockSize;
		dim3 inputGrid = ((numSamples * inputDim + blockSize - 1) / blockSize);

		if (sigmoid) {
			hipMalloc((void**)&dev_doutLinear, outputDim * numSamples * sizeof(float));
			checkCUDAError("cuda Malloc dev_din in failed");
			//Get derivative of softmax, and update 
			kern_dSigmoid<<<outputGrid, blockSize >>>(dev_dout, dev_doutLinear, inputDim, outputDim);
		}
		else {
			dev_doutLinear = dev_dout;
		}

		//Use matrix to compute dIn 
		kern_dIn<<<inputGrid, blockSize >>>(dev_doutLinear, W, dev_din, inputDim, outputDim, numSamples);


		//Update dw and db
		kern_dW<<<weightBiasGrid, blockSize >>>(W, b, dev_doutLinear, dev_in, inputDim, outputDim, numSamples, lr);

		//DEBUG STUFF
		float *myW= new float[inputDim * outputDim];
		hipMemcpy(myW, W, inputDim * outputDim * sizeof(float), hipMemcpyDeviceToHost);
		printf("MY WSTARTS\n");
		printFloatArray(myW, inputDim * outputDim);
		printf("MY WENDS\n");

		//Memcpy back the din info
		float *din = new float[inputDim * numSamples];
		hipMemcpy(din, dev_din, inputDim * numSamples * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("cuda Memcpy din in failed");


		//Free Mems
		hipFree(dev_doutLinear);
		hipFree(dev_din);
		return din;
	}

	void cpu_softmax(float *pred, int numSamples, int outputDim) {
		float rowSum = 0;
		for (int i = 0; i < numSamples; ++i) {
			rowSum = 0;
			for (int j = 0; j < outputDim; ++j) {
				pred[i * outputDim + j] = exp(pred[i * outputDim + j]);
				rowSum += pred[i * outputDim + j];
			}
			for (int k = 0; k < outputDim; ++k){
				pred[i * outputDim + k] /= rowSum;
			}
		}
	}

	float cpu_crossEntropy(float *pred, float *target, int numSamples, int outputDim, float* dout){
		float* log_likelihood = new float[numSamples];
		float llsum = 0;
		for (int i = 0; i < numSamples; ++i) {
			for (int c = 0; c < outputDim; ++c) {
				float ting = pred[i * outputDim + c];
				dout[i * outputDim + c] = ting;
			}
		}

		for (int i = 0; i < numSamples; ++i) {
			int offset = target[i];
			float ting = pred[i * outputDim + offset];
			log_likelihood[i] = -log(ting);
			llsum += -log(ting);
			dout[i * outputDim + offset] -= 1;
			for (int c = 0; c < outputDim; ++c) {
				dout[i * outputDim + c] /= numSamples;
			}
		}
		return llsum / numSamples;
	}

	float softmax_loss(float *pred, float *target, float *dout, int numSamples, int outputDim) {
		/* Returns a float representing the loss, and updates dout
		pred: Shape numSamples x outputDim
		target: Shape numSamples
		dout: Each element
		*/

		//Apply Softmax to pred
		cpu_softmax(pred, numSamples, outputDim);

		float loss = cpu_crossEntropy(pred, target, numSamples, outputDim, dout);
		return loss;
	}

	void XORTest() {
		//Network Structure
		int numSamples = 4;
		int inputDim = 2;
		int hiddenDim[1] = { 3 };
		int outputDim = 2;

		//XOR Input Array and Target Array
		float *x = new float[numSamples * inputDim];
		float *target = new float[numSamples * outputDim];
		x[0] = 0;
		x[1] = 0;
		target[0] = 0;
		x[2] = 0;
		x[3] = 1;
		target[1] = 1;
		x[4] = 1;
		x[5] = 0;
		target[2] = 1;
		x[6] = 1;
		x[7] = 1;
		target[3] = 0;

		//Build Layers
		AffineLayer* layer1 = new AffineLayer(inputDim, hiddenDim[0], numSamples);
		//layer1->setSigmoid(false);
		AffineLayer* layer1copy = new AffineLayer(inputDim, hiddenDim[0], numSamples);
		//layer1copy->setSigmoid(false);
		AffineLayer* layer2 = new AffineLayer(hiddenDim[0], outputDim, numSamples);
		layer2->setSigmoid(false);
		float lr = 0.01;
		for (int l = 0; l < 100; ++l) {
			/* FORWARD PROP */
			float *out0, *out1;
			printf("IN\n");
			printFloatArray(x, numSamples * outputDim);
			out0 = layer1->forward(x, numSamples);
			printf("OUT0\n");
			printFloatArray(out0, numSamples * outputDim);
			out1 = layer2->forward(out0, numSamples);
			printf("OUT1\n");
			printFloatArray(out1, numSamples * outputDim);

			/* CALCULATE LOSS */
			float *dout = new float[outputDim * numSamples];
			float loss = softmax_loss(out1, target, dout, numSamples, outputDim);
			printf("LOSS:%f\n", loss);
			printFloatArray(dout, outputDim * numSamples);

			/* BACKWARD PROP */
			float *dout1, *dout0;
			dout1 = layer2->backward(dout, lr);
			dout0 = layer1->backward(dout1, lr);
			printf("DOUT0\n");
			printFloatArray(dout0, inputDim * numSamples);
			printf("======================================\n", loss);
		}
	}
}
