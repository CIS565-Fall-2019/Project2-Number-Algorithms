#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define blockSize 128
#define blockWidth 16


namespace CharacterRecognition {
	using Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}

	//=====Initlialiations=======

	//layers
	double *dev_iLayer;
	double *dev_hLayer;
	double *dev_oLayer;

	double *dev_losses;
	double *dev_LossAvg;

	// gtruth and preds
	int *dev_gtruth;
	int *dev_preds;
	double * dev_preds_probab;

	//weights
	double *dev_w_kj;
	double *dev_w_ji;

	//Derivatives
	double *dev_dL_dw_ji;
	double *dev_dL_dw_kj;
	double *dev_dL_dscores;
	double *dev_dL_dscores_2;

	double *dev_hLayer_T;
	double *dev_iLayer_T;
	double *dev_w_ji_T;


	//=============================================
	// Rnadom Number Generation using cuRand on GPU
	//=============================================
	hiprandState *devState;

	__global__ void kernInitCurand(hiprandState *state, int N, unsigned long seed) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < N) {
			hiprand_init(seed, tid, 0, &state[tid]);
		}
	}

	__global__ void KernGenRand(hiprandState *state, int N, double *w) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < N) {
			w[tid] = (2.0*hiprand_uniform(&state[tid]) - 1.0); // Between -1 and 1
		}
	}

	//===================================================================
	//=====KERNEL DEFNITIONS FOR Forward and Backward====================
	//===================================================================


	void printArray(int n, int *a, bool abridged = false) {
		printf("    [ ");
		for (int i = 0; i < n; i++) {
			if (abridged && i + 2 == 15 && n > 16) {
				i = n - 2;
				printf("... ");
			}
			printf("%3d ", a[i]);
		}
		printf("]\n");
	}
	void printFloatArray(int n, double *a, bool abridged = false) {
		printf("    [ ");
		for (int i = 0; i < n; i++) {
			if (abridged && i + 2 == 15 && n > 16) {
				i = n - 2;
				printf("... ");
			}
			printf("%0.2f ", a[i]);
		}
		printf("]\n");
	}


	// Kernel for Gradient update on Weights
	__global__ void kernUpdateWeights(int N, double *dev_dw, double *dev_w, double LR) {

		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			dev_w[tid] = dev_w[tid] - (LR * dev_dw[tid]);
		}
	}

	// Kernel for derivative of sigmoid
	__global__ void kernGradSigmoid(int N, int H, double *dev_hLayer) {

		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N*H) {
			dev_hLayer[tid] = dev_hLayer[tid] * (1 - dev_hLayer[tid]);
		}
	}

	// Matrix Transpose
	__global__ void kernMatrixTranspose(int rows, int cols, double *matrix, double *matrix_T) {

		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int idx = blockIdx.x * blockDim.x + threadIdx.x;

		if (idx < cols && idy < rows) {
			int pos = idy * cols + idx;
			int tpos = idx * rows + idy;

			matrix_T[tpos] = matrix[pos];
		}
	}

	// Divide by N
	__global__ void kernDivNdscores(int N, int C, double *dev_dL_dscores) {

		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < N*C) {
			dev_dL_dscores[tid] /= N;
		}
	}

	// Compute dscores gradient
	__global__ void kernSetdscores(int N, int C, double *dev_dL_dscores, int *dev_gtruth) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			dev_dL_dscores[tid*C + dev_gtruth[tid]] -= 1;
		}
	}

	// compute predictions
	__global__ void kernPredsN(int N, int C, double* dev_oLayer, int* dev_gtruth, int* dev_preds, double * dev_preds_probab) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			dev_preds[tid] = dev_oLayer[tid*C + dev_gtruth[tid]] > 0.5 ? dev_gtruth[tid] : (dev_gtruth[tid] == 0 ? 1 : 0);
			dev_preds_probab[tid] = dev_oLayer[tid*C + dev_gtruth[tid]];
		}
	}

	// compute loss per example
	__global__ void kernLossPerN(int N, int C, double* dev_oLayer, int* dev_gtruth, double* dev_losses) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid < N) {
			dev_losses[tid] = -log(dev_oLayer[tid*C + dev_gtruth[tid]]);
		}
	}

	// kernel to compute exp softmax
	__global__ void kernSoftmax(int N, int C, double* scores) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (tid < N) {
			double sums = 0.0;

			for (int i = 0; i < C; i++) {
				sums += exp(scores[tid*C + i]);
			}

			for (int i = 0; i < C; i++) {
				scores[tid*C + i] = exp(scores[tid*C + i]) / sums;
			}
		}
	}

	// kern for sigmoid // f(x) = 1/(1 + e^-x).
	__global__ void kernSigmoid(int N, double *idata) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;

		if (tid < N) {
			idata[tid] = 1.0 / (1.0 + exp(-1*idata[tid]));
		}
	}

	// kern for element wise product 
	__global__ void kernElementProduct(int N, double *matrixA, double* matrixB) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;

		if (tid < N) {
			matrixA[tid] = matrixA[tid] * matrixB[tid];
		}
	}


	// kernel to to matmul // A mxn // B nxk // C mxk
	__global__ void kernMatrixMultiply(const double *dev_A, const double *dev_B, double *dev_C, int m, int n, int k) {

		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;

		double sum = 0;
		if (col < k && row < m)
		{
			for (int i = 0; i < n; i++)
				sum += dev_A[row * n + i] * dev_B[i * k + col];
			dev_C[row * k + col] = sum;
		}
	}

	// Dumb reduction
	__global__ void kernReduction(int N, double *dev_losses, double *dev_LossAvg) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		double sum = 0.0;
		if (tid == 0) {
			for (int i = 0; i < N; i++) {
				sum += dev_losses[i];
			}
			dev_LossAvg[0] = sum / N;
		}

	}

	// Ele wise addition A = A+B
	__global__ void kernAddition(int N, double *dev_A, double *dev_B) {

		int tid = blockIdx.x * blockDim.x + threadIdx.x;

		if (tid < N) {
			dev_A[tid] += dev_B[tid];
		}

	}

	void trainMLP(int N, int D, int H, int C, double *idata, int *preds, int *gtruth, int epochs, 
		          double *lossAvgPerEpoch, const double LR, double *w1, double *w2, unsigned long seed) {

		timer().startGpuTimer();

		// N = number of examples
		// D = dim of each example 
		// H = Hidden state nodes
		// C = number of classes

		// NETWORK DEFITION_____________
		// Compute f1		= W1*X1
		// Compute X2		= Sig(f1)
		// Compute Scroes S = W2*X2
		// Compute Probab P = Softmax(S)
		// Compute Loss   L = CEntropy(P)

		//================================================================
		//======================INITIALIZATIONS===========================
		//================================================================

		// Allocate input layer
		hipMalloc((void**)&dev_iLayer, N*D * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_iLayer failed!");

		hipMemcpy(dev_iLayer, idata, N*D * sizeof(double), hipMemcpyHostToDevice);
		checkCUDAErrorFn("hipMemcpyToSymbol from idata to dev_iLayer failed!");


		// Allocate hidden layer
		hipMalloc((void**)&dev_hLayer, N*H* sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer failed!");


		// Allocate output layer
		hipMalloc((void**)&dev_oLayer, N*C* sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_oLayer failed!");


		// Allocate losses holder
		hipMalloc((void**)&dev_losses, N * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_losses failed!");

		hipMalloc((void**)&dev_LossAvg, 1*sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_LossAvg failed!");


		// Allocate gtruth and preds
		hipMalloc((void**)&dev_gtruth, N * sizeof(int));
		checkCUDAErrorFn("hipMalloc dev_gtruth failed!");

		hipMemcpy(dev_gtruth, gtruth, N * sizeof(int), hipMemcpyHostToDevice);
		checkCUDAErrorFn("hipMemcpyToSymbol from gtruth to dev_gtruth failed!");

		hipMalloc((void**)&dev_preds, N * sizeof(int));
		checkCUDAErrorFn("hipMalloc dev_preds failed!");

		hipMalloc((void**)&dev_preds_probab, N * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_preds_probab failed!");

		// Allocate Weights
		hipMalloc((void**)&dev_w_kj, D*H * sizeof(double)); //w1
		checkCUDAErrorFn("hipMalloc dev_w_kj failed!");

		hipMalloc((void**)&dev_w_ji, C*H * sizeof(double)); //w2
		checkCUDAErrorFn("hipMalloc dev_w_ji failed!");


		// Allocate Derivatives
		hipMalloc((void**)&dev_dL_dw_kj, D*H * sizeof(double)); //dw1
		checkCUDAErrorFn("hipMalloc dev_w_kj failed!");

		hipMalloc((void**)&dev_dL_dw_ji, C*H * sizeof(double)); //dw2
		checkCUDAErrorFn("hipMalloc dev_w_ji failed!");

		hipMalloc((void**)&dev_dL_dscores, N*C * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_dL_dscores failed!");

		hipMalloc((void**)&dev_dL_dscores_2, N*C * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_dL_dscores_2 failed!");


		// Allocate transposes
		hipMalloc((void**)&dev_hLayer_T, N*H * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");

		hipMalloc((void**)&dev_iLayer_T, N*D * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_hLayer_T failed!");

		hipMalloc((void**)&dev_w_ji_T, C*H * sizeof(double));
		checkCUDAErrorFn("hipMalloc dev_w_ji_T failed!");

		//==============================
		// Initialise Weights
		//==============================
		hipMalloc((void**)&devState, H*D * sizeof(hiprandState));

		kernInitCurand << <((D*H + blockSize - 1) / blockSize), blockSize >> > (devState, D*H, seed);
		checkCUDAErrorFn("KernInitCurand failed!");
		KernGenRand << <((D*H + blockSize - 1) / blockSize), blockSize >> > (devState, D*H, dev_w_kj);//w1
		checkCUDAErrorFn("KernGenRand dev_w_kj failed!");

		kernInitCurand << <((H*C + blockSize - 1) / blockSize), blockSize >> > (devState, H*C, seed); 
		checkCUDAErrorFn("KernInitCurand failed!");
		KernGenRand << <((H*C + blockSize - 1) / blockSize), blockSize >> > (devState, H*C, dev_w_ji);//w2
		checkCUDAErrorFn("KernGenRand dev_w_kj failed!");


		//================================================================
		//======================TRAINING LOOP=============================
		//================================================================
		double *tmp = new double[N*D];
		double *tmp2 = new double[N*D];
		double *lossesN = new double[N];
		printf("--------------------------------------------\n");
		printf("One Hidden Layer MLP | Configuration \n");
		printf("--------------------------------------------\n");
		printf("Number of Examples            | N = %d \n",N);
		printf("Dimensionality of each Example| D = %d \n",D);
		printf("Number of Hidden Layer Nodes  | H = %d \n",H);
		printf("Total Number of Classes       | C = %d \n",C);
		printf("Activation    = Sigmoid \n");
		printf("Loss Function = Cross Entropy \n");
		printf("--------------------------------------------\n");

		//printf("\nInput DATA  ");
		//printf("\nInput DATA  ");
		//printFloatArray(N*D, idata, true);
		dim3 dimBlock(blockWidth, blockWidth);
		dim3 dimGrid;

		for (int i = 0; i < epochs; i++) {

			//================================================================
			//========================= FORWARD ==============================

			// STEP 1
			// f1 = W1*X1 (Matrix Mul)
			//=================================
			// dev_hLayer = dev_iLayer*dev_w_kj 
			//   NxH      =    NxD         DxH 
			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_iLayer, dev_w_kj, dev_hLayer, N, D, H);

			// STEP 2
			// X2         = Sigmoid(f1) 
			// dev_hLayer = sigmoid(dev_hLayer)
			//   NxH     =    NxH 
			kernSigmoid << <((N*H + blockSize - 1) / blockSize), blockSize >> > (N*H, dev_hLayer);


			// STEP 3
			// Scores S = W2*X2 (Matrix Mul)
			// dev_oLayer = dev_hLayer*dev_w_ji 
			//   NxC      =    NxH         HxC
			dimGrid.x = (C + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_hLayer, dev_w_ji, dev_oLayer, N, H, C);
			checkCUDAErrorFn("kernMatrixMultiply failed!");


			// STEP 4
			// P = Softmax(S) 
			// dev_smaxDen = Sum_Over_classses(dev_olayer)
			// dev_olayer = dev_olayer/Sum_Over_classses
			//   NxC      =    NxC         1
			kernSoftmax << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_oLayer);
			checkCUDAErrorFn("kernSoftmax failed!");

			// STEP 5
			// Compute Losses | Cross Entropy Loss
			kernLossPerN << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_oLayer, dev_gtruth, dev_losses);
			checkCUDAErrorFn("kernLossPerN  failed!");

			// Cpoy loss to CPU
			//hipMemcpy(lossesN, dev_losses, N * sizeof(double), hipMemcpyDeviceToHost);
			//checkCUDAErrorFn("hipMemcpyFromSymbol from dev_losses to lossesN failed!");
			//printf("Post dev_losses [Loss = CEntropy(P)]\n");
			//printFloatArray(N, lossesN, true);


			// Compute Predictions
			kernPredsN << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_oLayer, dev_gtruth, dev_preds, dev_preds_probab);
			hipMemcpy(preds, dev_preds, N * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyDeviceToHost from dev_preds to preds failed!");
			hipMemcpy(tmp2, dev_preds_probab, N * sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyDeviceToHost from dev_preds_probab to tmp failed!");


			// STEP 5.2
			// Compute Avg of Losses
			kernReduction << <((N + blockSize - 1) / blockSize), blockSize >> > (N, dev_losses, dev_LossAvg);
			// Copy back to cpu
			hipMemcpy(lossAvgPerEpoch + i, dev_LossAvg, sizeof(double), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpyFromSymbol from dev_LossAvg to tmp failed!");
			
			if (i % 1000 == 0) {
				printf("Epoch : %3d | LossAvg %3f \n", i, lossAvgPerEpoch[i]);
				printf("GroundTruth :");
				printArray(N, gtruth, true);
				printf("Predictions :");
				printArray(N, preds, true);
				printf("Confidence  :");
				printFloatArray(N, tmp2, true);
				printf("\n");
			}

			//=================================================================
			//========================= BACKPROP ==============================

			//===============================
			// STEP 1 : Gradient wrt w_kj W2
			//===============================
			// dW_ji = Probs_k - [1](gth == k) dev_dL_dscores;

			hipMemcpy(dev_dL_dscores, dev_oLayer, N*C * sizeof(double), hipMemcpyDeviceToDevice);
			checkCUDAErrorFn("hipMemcpyFromSymbol from probabs to dev_dL_dscores failed!");

			kernSetdscores << <((N + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_dL_dscores, dev_gtruth);
			checkCUDAErrorFn("kernSetdscores failed!");

			kernDivNdscores << <((N*C + blockSize - 1) / blockSize), blockSize >> > (N, C, dev_dL_dscores);
			checkCUDAErrorFn("kernDivNdscores failed!");

			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixTranspose << <dimGrid, dimBlock >> > (N, H, dev_hLayer, dev_hLayer_T);

			dimGrid.x = (C + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (H + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_hLayer_T, dev_dL_dscores, dev_dL_dw_ji, H, N, C);
			checkCUDAErrorFn("kernMatrixMultiply for dev_dL_dw_ji failed!");

			//===============================
			// STEP 2 : Gradient wrt w_kj W1
			//===============================

			// Transpose Wji (W2)
			dimGrid.x = (C + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (H + dimBlock.y - 1) / dimBlock.y;
			kernMatrixTranspose << <dimGrid, dimBlock >> > (H, C, dev_w_ji, dev_w_ji_T);

			// Transpose Input Data
			dimGrid.x = (D + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixTranspose << <dimGrid, dimBlock >> > (N, D, dev_iLayer, dev_iLayer_T);

			// Mul dev_dL_dscores * dev_w_kj_T == dev_dL_dscores_2
			//             NxC          CxH             NxH
			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_dL_dscores, dev_w_ji_T, dev_dL_dscores_2, N, C, H);
			checkCUDAErrorFn("kernMatrixMultiply for dev_dL_dscores_2 failed!");

			// compute sig gradient on dev_hlayer N*H [IN PLACE]
			kernGradSigmoid << <((N*H + blockSize - 1) / blockSize), blockSize >> > (N, H, dev_hLayer);
			checkCUDAErrorFn("kernGradSigmoid failed!");

			//Element wise mul dev_dL_dscores_2 [INPLACE] = dev_dL_dscores_2 . dev_hlayer[sig gradient] 
			kernElementProduct << <((N*H + blockSize - 1) / blockSize), blockSize >> > (N*H, dev_dL_dscores_2, dev_hLayer);
			checkCUDAErrorFn("kernElementProduct failed!");

			// matrix Mul final with Xi_T
			dimGrid.x = (H + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (D + dimBlock.y - 1) / dimBlock.y;
			kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_iLayer_T, dev_dL_dscores_2, dev_dL_dw_kj, D, N, H);
			checkCUDAErrorFn("kernMatrixMultiply for dev_dL_dw_kj failed!");


			//=================================================================
			// STEP 3 : Update Weights ========================================
			//=================================================================

			// Update weights kj W1
			kernUpdateWeights << <((D*H + blockSize - 1) / blockSize), blockSize >> > (D*H, dev_dL_dw_kj, dev_w_kj, LR);
			checkCUDAErrorFn("kernUpdateWeights dev_w_kj failed!");

			// InitUpdate weights ji W2
			kernUpdateWeights << <((H*C + blockSize - 1) / blockSize), blockSize >> > (H*C, dev_dL_dw_ji, dev_w_ji, LR);
			checkCUDAErrorFn("kernUpdateWeights dev_w_ji failed!");

			//printf("\n-----------------------------------------------------\n\n");
		}


		printf("Finished training.\n");
		float count = 0.0;
		for (int n = 0; n < N; n++) {
			if (preds[n] == gtruth[n]) {
				count += 1;
			}
		}
		float acc = count / N;
		printf("Accuracy: %0.2f Percent \n", acc*100.0);

		// SAVE WEIGHTS
		hipMemcpy(w1, dev_w_kj, H*D*sizeof(double), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpyFromSymbol from  dev_w_kj to w1 failed!");

		hipMemcpy(w2, dev_w_ji, H*C*sizeof(double), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpyFromSymbol from  dev_w_ji to w2 failed!");

		//printf("losses:\n");
		//printFloatArray(epochs, lossAvgPerEpoch, true);

		//====================
		// CleanUp
		//====================
		hipFree(dev_iLayer);
		hipFree(dev_hLayer);
		hipFree(dev_oLayer);

		hipFree(dev_losses);

		hipFree(dev_gtruth);
		hipFree(dev_preds);
		hipFree(dev_preds_probab);

		hipFree(dev_w_kj);
		hipFree(dev_w_ji);

		hipFree(dev_dL_dw_ji);
		hipFree(dev_dL_dw_kj);

		hipFree(dev_dL_dscores);
		hipFree(dev_dL_dscores_2);

		hipFree(dev_hLayer_T);
		hipFree(dev_iLayer_T);

		delete(tmp);
		delete(tmp2);
		delete(lossesN);

		timer().endGpuTimer();
	}
}
