#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"
#include <hipblas.h>
#include <hiprand.h>
#include<memory.h>
#include<iostream>

#define blockSize 128

hipblasHandle_t handle;
namespace CharacterRecognition {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	__global__ void kernAddVectors(int n, float* g, float* bias, float* result) {
		int index = (blockIdx.x*blockDim.x) + threadIdx.x;
		if (index >= n)
			return;
		result[index] = g[index] + bias[index];
	}

	__global__ void kernUpdateParameters(int n, float *input, float *grad, float alpha) {
		int index = (blockIdx.x*blockDim.x) + threadIdx.x;
		if (index >= n)
			return;
		input[index] = input[index] - alpha * grad[index];
	}

	__global__ void kernSubVectors(int n, float* y, float* yhat, float* result) {
		int index = (blockIdx.x*blockDim.x) + threadIdx.x;
		if (index >= n)
			return;
		result[index] = yhat[index] - y[index];
	}

	__global__ void kernInitBiasVectors(int n, float* b, float value) {
		int index = (blockIdx.x*blockDim.x) + threadIdx.x;
		if (index >= n)
			return;
		b[index] = value;
	}

	__global__ void kernUpSweep(int n, int d, float *itemp) {
		int k = (blockIdx.x*blockDim.x) + threadIdx.x;
		if (k > (n - 1)) {
			return;
		}
		int power = 1 << (d + 1);
		int power_2 = 1 << d;
		if (k % power == 0 && k + power - 1 < n && k + power_2 - 1 < n)
			itemp[k + power - 1] += itemp[k + power_2 - 1];
	}

	__global__ void kernSoftmaxActivation(int n, float *g, float *output, float exp_sum) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index >= n)
			return;
		output[index] = expf(g[index]) / exp_sum;
	}

	__global__ void kernSoftmaxDerivative(int n, float *input, float *grad, float exp_sum) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index >= n)
			return;
		grad[index] = (exp_sum*expf(input[index]) - expf(input[index]) * expf(input[index])) / (exp_sum * exp_sum);
	}

	__global__ void kernReluActivationForward(int n, float* g, float* a) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index >= n)
			return; 
		a[index] = fmaxf(g[index], 0);
	}

	__global__ void kernReluDerivative(int n, float* input, float* grad) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;

		if (index >= n)
			return;
		grad[index] = (input[index] > 0) ? 1 : 0;
	}

	__global__ void kernCopyVectors(int n, float *g, float *output) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= n)
			return;
		output[index] = expf(g[index]);
	}

	__global__ void kernDerivativeLoss(int n, float *y, float *y_pred, float *output) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= n) {
			return;
		}
		output[index] = -y[index] / y_pred[index] + (1 - y[index]) / (1 - y_pred[index]);
	}
	__global__ void kernElementWiseMultiplication(int n, float *input1, float *input2, float *output) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= n) {
			return;
		}
		output[index] = input1[index] * input2[index];
	}

	void random_init(float * A, int rows, int cols) {
		hiprandGenerator_t prng;
		hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
		hiprandGenerateNormal(prng, A, rows * cols, 0, 2.0/rows);
	}

	//C(m,n) = A(m,k)*B(k,n)
	void mmul(const float* A, const float* B, float* C, const int m, const int k, const int n, int a_trans_flag, int b_trans_flag
		, int lda, int ldb, int ldc) {
		const float alf = 1;
		const float bet = 0;
		const float *alpha = &alf;
		const float *beta = &bet;
		std::cout << a_trans_flag << " " << b_trans_flag << std::endl;
		if(a_trans_flag == 0 && b_trans_flag == 0)
			hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
		else if(a_trans_flag == 0 && b_trans_flag == 1)
			hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
		else if(a_trans_flag == 1 && b_trans_flag == 0)
			hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	}

	void printCuda(float *a1, int n, std::string name) {
		float *print_a = new float[n];
		std::cout << name.c_str() << std::endl;
		std::cout << "{" << std::endl;
		hipMemcpy(print_a, a1, n * sizeof(float), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++) {
			std::cout << "\t" << print_a[i] << std::endl;
		}
		std::cout << "}" << std::endl;
		delete[]print_a;
	}

	NeuralNet::NeuralNet(int input_size, int classes, vector<int>layers) {
		layer_sizes.push_back(input_size);

		// Set all layer sizes
		for (int i = 0; i < layers.size(); i++)
			layer_sizes.push_back(layers[i]);
		layer_sizes.push_back(classes);
		// Temporary variables to be pushed;
		float *z_t, *dz_t, *a_t, *da_t, *w_t, *dw_t, *b_t, *db_t, *ghat_t;
		// Some dummy mallocs to be pushed for the 0th(input) layer
		// We treat a0 as the input layer.
		hipMalloc((void**)&z_t, sizeof(float));
		checkCUDAError("Cuda Malloc for z failed.");
		z.push_back(z_t);

		hipMalloc((void**)&dz_t, sizeof(float));
		checkCUDAError("Cuda Malloc for dz failed.");
		dz.push_back(dz_t);

		hipMalloc((void**)&a_t, layer_sizes[0] * 1 * sizeof(float));
		checkCUDAError("Cuda Malloc for a failed.");
		a.push_back(a_t);

		hipMalloc((void**)&da_t, layer_sizes[0] * 1 * sizeof(float));
		checkCUDAError("Cuda Malloc for da failed.");
		da.push_back(da_t);

		hipMalloc((void**)&w_t, sizeof(float));
		checkCUDAError("Cuda Malloc for weights failed.");
		w.push_back(w_t);

		hipMalloc((void**)&dw_t, sizeof(float));	
		checkCUDAError("Cuda Malloc for derivative of weights failed.");
		dw.push_back(dw_t);
	
		hipMalloc((void**)&b_t, sizeof(float));
		checkCUDAError("Cuda Malloc for bias failed.");
		b.push_back(b_t);

		hipMalloc((void**)&db_t, sizeof(float));
		checkCUDAError("Cuda Malloc for derivatives of bias failed.");
		db.push_back(db_t);

		hipMalloc((void**)&ghat_t, sizeof(float));
		checkCUDAError("Cuda Malloc for derivatives of bias failed.");
		ghat.push_back(ghat_t);


		// The following loop allocates sizes to all the weights, bias, a and z vectors and their gradients.
		for (int i = 1; i < layer_sizes.size(); i++) {

			hipMalloc((void**)&z_t, layer_sizes[i] * 1 * sizeof(float));
			checkCUDAError("Cuda Malloc for z failed.");
			z.push_back(z_t);

			hipMalloc((void**)&dz_t, layer_sizes[i] * 1 * sizeof(float));
			checkCUDAError("Cuda Malloc for dz failed.");
			dz.push_back(dz_t);

			hipMalloc((void**)&a_t, layer_sizes[i] * 1 * sizeof(float));
			checkCUDAError("Cuda Malloc for a failed.");
			a.push_back(a_t);

			hipMalloc((void**)&da_t, layer_sizes[i] * 1 * sizeof(float));
			checkCUDAError("Cuda Malloc for da failed.");
			da.push_back(da_t);

			hipMalloc((void**)&w_t, layer_sizes[i] * layer_sizes[i - 1] * sizeof(float));
			checkCUDAError("Cuda Malloc for weights failed.");
			w.push_back(w_t);

			hipMalloc((void**)&dw_t, layer_sizes[i] * layer_sizes[i - 1] * sizeof(float));
			checkCUDAError("Cuda Malloc for derivative of weights failed.");
			dw.push_back(dw_t);

			hipMalloc((void**)&b_t, layer_sizes[i] * 1 * sizeof(float));
			checkCUDAError("Cuda Malloc for bias failed.");
			b.push_back(b_t);

			hipMalloc((void**)&db_t, layer_sizes[i] * 1 * sizeof(float));
			checkCUDAError("Cuda Malloc for derivatives of bias failed.");
			db.push_back(db_t);

			hipMalloc((void**)&ghat_t, layer_sizes[i] * 1 * sizeof(float));
			checkCUDAError("Cuda Malloc for derivatives of activations failed");
			ghat.push_back(ghat_t);
			
		}
		// Avoid those memory leaks :)
		hipFree(z_t);
		hipFree(dz_t);
		hipFree(a_t);
		hipFree(da_t);
		hipFree(w_t);
		hipFree(dw_t);
		hipFree(b_t);
		hipFree(db_t);
		hipFree(ghat_t);

		dim3 fullBlocksPerGrid;
		// The following for loop initializes weights according to normal distribution 
		// We are using he-normal initialization here because of ReLU activation function
		for (int i = 1; i < layer_sizes.size(); i++) {
		//	fullBlocksPerGrid = ((layer_sizes[i]*layer_sizes[i-1] + blockSize - 1) / blockSize);
		//	kernInitBiasVectors << <fullBlocksPerGrid, blockSize >> > (layer_sizes[i] * layer_sizes[i-1] , w[i], 0);
			random_init(w[i], layer_sizes[i], layer_sizes[i - 1]);
		}
		// The following loop initializes the bias to a small value.
		// It invokes a kernel which fills the bias vector with the desired value
		for (int i = 1; i < layer_sizes.size(); i++) {
			fullBlocksPerGrid = ((layer_sizes[i] + blockSize - 1) / blockSize);
			kernInitBiasVectors << <fullBlocksPerGrid, blockSize >> > (layer_sizes[i], b[i], 0.1);
		}
		// Create a cublas handle for matrix multiplication
		hipblasCreate(&handle);
	}

	
	float* NeuralNet::forward(float *input) {

		// a^[0] will be the input
		hipMemcpy(a[0], input, layer_sizes[0] * sizeof(float), hipMemcpyHostToDevice);
		// The activation for every layer but the last is relu, so the steps will be the same.
		// The equations here are
		//z[l] = w[l]a[l-1] + b[l]
		// a[l] = relu(z[l])
		dim3 fullBlocksPerGrid;
		int L = layer_sizes.size() - 1;
		for (int i = 1; i < L; i++) {
			// Do the matrix multiplication to find w[l]a[l-1] and store in z[l]
			mmul(w[i], a[i - 1], z[i], layer_sizes[i], layer_sizes[i - 1], 1, 0,0,layer_sizes[i], layer_sizes[i-1],layer_sizes[i]);
			// Add the bias vector to it
			fullBlocksPerGrid = ((layer_sizes[i] + blockSize - 1) / blockSize);
			kernAddVectors << <fullBlocksPerGrid, blockSize >> > (layer_sizes[i], b[i], z[i], z[i]);
			// Apply the Relu activation function
			kernReluActivationForward << <fullBlocksPerGrid, blockSize >> > (layer_sizes[i], z[i], a[i]);
		}
		// Now the softmax output for the final layer which will give the probability of belonging to each class
		// We will first calculate the z for the final layer
		mmul(w[L], a[L - 1], z[L], layer_sizes[L], layer_sizes[L - 1], 1,0,0,layer_sizes[L], layer_sizes[L-1], layer_sizes[L]);
		fullBlocksPerGrid = ((layer_sizes[L] + blockSize - 1) / blockSize);
		kernAddVectors << <fullBlocksPerGrid, blockSize >> > (layer_sizes[L], b[L], z[L], z[L]);
		// We will then calculate the sum(e^(z[L]))
		// Doing it on the CPU because in the stream compaction code, the cpu implementation was faster for smaller inputs.
		float *y_pred = new float[layer_sizes[L]];
		hipMemcpy(y_pred, z[L], layer_sizes[L] * sizeof(float), hipMemcpyDeviceToHost);
		float exp_sum = 0;
		for (int i = 0; i < layer_sizes[L]; i++) {
			exp_sum += expf(y_pred[i]);
		}
		// Now apply softmax activation

		fullBlocksPerGrid = ((layer_sizes[L] + blockSize - 1) / blockSize);
		
		kernSoftmaxActivation << <fullBlocksPerGrid, blockSize >> > (layer_sizes[L], z[L], a[L], exp_sum);
		hipMemcpy(y_pred, a[L], layer_sizes[L] * sizeof(float), hipMemcpyDeviceToHost);
		return y_pred;
		
	}
	void NeuralNet::backward(float *y) {
		int L = layer_sizes.size() - 1;
		// We will first populate da[L] as the derivative of loss with respect to y_pred.
		float *y_cuda;
		hipMalloc((void**)&y_cuda, layer_sizes[L] * sizeof(float));
		hipMemcpy(y_cuda, y, layer_sizes[L] * sizeof(float), hipMemcpyHostToDevice);
		dim3 fullBlocksPerGrid;
		fullBlocksPerGrid = ((layer_sizes[L] + blockSize - 1) / blockSize);
		kernDerivativeLoss<<<fullBlocksPerGrid, blockSize>>>(layer_sizes[L], y_cuda, a[L], da[L]);
		// The equations for the backpropagation are
		// dz[l] = da[l]*g'[l](z[l]) where * means element wise
		// dw[l] = dz[l]a[l-1].T
		// db[l] = dz[l]
		// da[l-1] = W[l].Tdz[l]
		// Now the softmax derivative for the last but one layer
		float *sum_cp = new float[layer_sizes[L]];
		hipMemcpy(sum_cp, z[L], layer_sizes[L] * sizeof(float), hipMemcpyDeviceToHost);
		float exp_sum = 0;
		for (int i = 0; i < layer_sizes[L]; i++) {
			exp_sum += expf(sum_cp[i]);
		}
		kernSoftmaxDerivative << <fullBlocksPerGrid, blockSize >> > (layer_sizes[L], z[L], ghat[L], exp_sum);
		kernElementWiseMultiplication << <fullBlocksPerGrid, blockSize >> > (layer_sizes[L], da[L], ghat[L], dz[L]);
		// dw[l] = dz[l]a[l-1].T
		mmul(dz[L], a[L - 1], dw[L], layer_sizes[L], 1, layer_sizes[L-1], 0, 1,layer_sizes[L], layer_sizes[L-1], layer_sizes[L]);
		//db[l] = dz[l]
		hipMemcpy(db[L], dz[L], layer_sizes[L] * sizeof(float), hipMemcpyDeviceToDevice);
		//da[l - 1] = W[l].Tdz[l]
		mmul(w[L], dz[L], da[L - 1], layer_sizes[L - 1], layer_sizes[L], 1, 1, 0, layer_sizes[L], layer_sizes[L], layer_sizes[L - 1]);
		//Now for the ReLU layers
		for (int i = L - 1; i >= 1; i--) {
			kernReluDerivative << <fullBlocksPerGrid, blockSize >> > (layer_sizes[i], z[i], ghat[i]);
			kernElementWiseMultiplication << <fullBlocksPerGrid, blockSize >> > (layer_sizes[i], da[i], ghat[i], dz[i]);
			mmul(dz[i], a[i - 1], dw[i], layer_sizes[i], 1, layer_sizes[i - 1], 0, 1, layer_sizes[i], layer_sizes[i - 1], layer_sizes[i]);
			hipMemcpy(db[i], dz[i], layer_sizes[i] * sizeof(float), hipMemcpyDeviceToDevice);
			mmul(w[i], dz[i], da[i - 1], layer_sizes[i - 1], layer_sizes[i], 1, 1, 0, layer_sizes[i], layer_sizes[i], layer_sizes[i - 1]);
		}
		// Now we will update the weights and bias
		for (int i = 1; i <= L; i++) {
			printCuda(w[i], layer_sizes[i] * layer_sizes[i - 1], "W");
			printCuda(dw[i], layer_sizes[i] * layer_sizes[i - 1], "dw");
			fullBlocksPerGrid = ((layer_sizes[i]*layer_sizes[i-1] + blockSize - 1) / blockSize);
			kernUpdateParameters <<< fullBlocksPerGrid, blockSize >> > (layer_sizes[i] * layer_sizes[i - 1], w[i], dw[i], 0.01);
			printCuda(w[i], layer_sizes[i] * layer_sizes[i - 1], "W_updated");
			fullBlocksPerGrid = ((layer_sizes[i] + blockSize - 1) / blockSize);
			kernUpdateParameters <<< fullBlocksPerGrid, blockSize >>> (layer_sizes[i], b[i], db[i], 0.01);
		}
		// Avoid the memory leaks
		hipFree(y_cuda);

	}
	NeuralNet::~NeuralNet() {
		// Here comes the destructor, will free those memories ...
		for (auto x : w)
			hipFree(x);
		for (auto x : dw)
			hipFree(x);
		for (auto x : b)
			hipFree(x);
		for (auto x : db)
			hipFree(x);
		for (auto x : z)
			hipFree(x);
		for (auto x : dz)
			hipFree(x);
		for (auto x : a)
			hipFree(x);
		for (auto x : da)
			hipFree(x);
		for (auto x : ghat)
			hipFree(x);

		
		hipblasDestroy(handle);
	}
    // TODO: __global__

    /**
        * Example of use case (follow how you did it in stream compaction)
        */
    /*void scan(int n, int *odata, const int *idata) {
        timer().startGpuTimer();
        // TODO
        timer().endGpuTimer();
    }
    */

	// TODO: implement required elements for MLP sections 1 and 2 here
}
