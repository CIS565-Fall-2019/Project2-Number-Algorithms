#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <hip/device_functions.h>
//#include ""
#include "common.h"
#include "mlp.h"
#include <vector>

/*! Block size used for CUDA kernel launch. */
#define blockSize 32
#define index(i,j,ld) (((j)*(ld))+(i))



namespace CharacterRecognition {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	typedef struct _matrixSize {
		int WA, HA, WB, HB, WC, HC;
	} sMatrixSize;

	void printMat(float*P, int uWP, int uHP) {
		int i, j;
		for (i = 0; i < uHP; i++) {
			printf("\n");
			for (j = 0; j < uWP; j++)
				printf("%f ", P[index(i, j, uHP)]);
		}
	}

	void randomInit(float *data, int size) {
		for (int i = 0; i < size; ++i)
			data[i] = rand() / (float)RAND_MAX;
	}

	void fixedInit(float *data, int size) {
		if (size == 4) {
			data[0] = 10.1f;
			data[1] = 0.9f;
			data[2] = 20.0f;
			data[3] = 0.87f;
		}
		else if (size == 2) {
			data[0] = 41.0f;
			data[1] = -54.0f;
		}
	}

	void indexInit(float *data, int size) {
		for (int i = 0; i < size; ++i)
			data[i] = (float)i;
	}

	int getNum(int &n, float *v) {
		// Generate a random number 
		srand(time(NULL));
		// Make sure the number is within the index range 
		int index = rand() % n;
		// Get random number from the vector 
		int num = v[index];
		// Remove the number from the vector 
		std::swap(v[index], v[n - 1]);
		n--;
		// Return the removed number 
		return num;
	}

	void generateRandom(int n, float *permuteData) {
		float *v = (float *)malloc(n);
		// Fill the vector with the values  1, 2, 3, ..., n 
		for (int i = 0; i < n; i++) {
			v[i] = i;
		}
		// While vector has elements get a random number from the vector and print it 
		int i = 0;
		while (n > 0) {
			permuteData[i] = getNum(n,v);
			i++;
		}
	}

	/*void deviceMemory(bool create = false, float *Xi = NULL, float *wI = NULL, float *wO = NULL, sMatrixSize &hidden_matrix_size = {}, sMatrixSize &output_matrix_size = {}, float *dev_X = NULL, float *dev_wI = NULL, float *dev_wO = NULL, float *dev_h1 = NULL, float *dev_pred = NULL) {
		if (create) {
			unsigned int size_X = hidden_matrix_size.WB * hidden_matrix_size.HB;
			unsigned int mem_size_X = sizeof(float) * size_X;
			unsigned int size_wI = hidden_matrix_size.WA * hidden_matrix_size.HA;
			unsigned int mem_size_wI = sizeof(float) * size_wI;
			unsigned int size_wO = output_matrix_size.WA * output_matrix_size.HA;
			unsigned int mem_size_wO = sizeof(float) * size_wO;
			unsigned int size_h1 = hidden_matrix_size.WC * hidden_matrix_size.HC;
			unsigned int mem_size_h1 = sizeof(float) * size_h1;
			unsigned int size_pred = output_matrix_size.WC * output_matrix_size.HC;
			unsigned int mem_size_pred = sizeof(float) * size_pred;

			hipMalloc((void **)&dev_X, mem_size_X);
			checkCUDAError("hipMalloc dev_X");
			hipMalloc((void **)&dev_wI, mem_size_wI);
			checkCUDAError("hipMalloc dev_wI");
			hipMalloc((void **)&dev_wO, mem_size_wO);
			checkCUDAError("hipMalloc dev_wO");
			hipMalloc((void **)&dev_h1, mem_size_h1);
			checkCUDAError("hipMalloc dev_h1");
			hipMalloc((void **)&dev_pred, mem_size_pred);
			checkCUDAError("hipMalloc dev_pred");
		}
		else {
			hipFree(dev_X);
			hipFree(dev_wI);
			hipFree(dev_wO);
			hipFree(dev_h1);
			hipFree(dev_pred);
		}
	}*/

	template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A, float *B, int wA, int wB) {
		// Block index
		int bx = blockIdx.x;
		int by = blockIdx.y;

		// Thread index
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		// Index of the first sub-matrix of A processed by the block
		int aBegin = wA * BLOCK_SIZE * by;

		// Index of the last sub-matrix of A processed by the block
		int aEnd = aBegin + wA - 1;

		// Step size used to iterate through the sub-matrices of A
		int aStep = BLOCK_SIZE;

		// Index of the first sub-matrix of B processed by the block
		int bBegin = BLOCK_SIZE * bx;

		// Step size used to iterate through the sub-matrices of B
		int bStep = BLOCK_SIZE * wB;

		// Csub is used to store the element of the block sub-matrix
		// that is computed by the thread
		float Csub = 0;

		// Loop over all the sub-matrices of A and B
		// required to compute the block sub-matrix
		for (int a = aBegin, b = bBegin;
			a <= aEnd;
			a += aStep, b += bStep) {
			// Declaration of the shared memory array As used to
			// store the sub-matrix of A
			__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

			// Declaration of the shared memory array Bs used to
			// store the sub-matrix of B
			__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

			// Load the matrices from device memory
			// to shared memory; each thread loads
			// one element of each matrix
			As[ty][tx] = A[a + wA * ty + tx];
			Bs[ty][tx] = B[b + wB * ty + tx];

			// Synchronize to make sure the matrices are loaded
			__syncthreads();

			// Multiply the two matrices together;
			// each thread computes one element
			// of the block sub-matrix
			#pragma unroll

			for (int k = 0; k < BLOCK_SIZE; ++k) {
				Csub += As[ty][k] * Bs[k][tx];
			}

			// Synchronize to make sure that the preceding
			// computation is done before loading two new
			// sub-matrices of A and B in the next iteration
			__syncthreads();
		}
	}

	void matrixMultiply(hipblasHandle_t* handle, sMatrixSize &matrix_size, float *d_A, float *d_B, float *d_C){
			const float alpha = 1.0f;
			const float beta = 0.0f;
			hipblasSgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.WB, matrix_size.HA, matrix_size.WA, &alpha, d_B, matrix_size.WB, d_A, matrix_size.WA, &beta, d_C, matrix_size.WB);
			checkCUDAError("matrix multiply");
	}

	// TODO: implement required elements for MLP sections 1 and 2 here
	__global__ void kernSigmoid(int n, float *input) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n) {
			return;
		}
		input[index] = 1.0f / (1 + exp(-input[index]));
	}

	__global__ void kernSigmoid2(int n, float *input) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n) {
			return;
		}
		input[index] = 1.0f / (1 + exp(-input[index]));
	}
	
	void backward(){}

	void forward(float *pred, float *Xi, float *wI, float *wO, sMatrixSize &hidden_matrix_size, sMatrixSize &output_matrix_size) {
		// allocate device memory
		float *dev_X, *dev_wI, *dev_wO, *dev_h1, *dev_pred;
		//deviceMemory(true, Xi, wI, wO, hidden_matrix_size, output_matrix_size, dev_X, dev_wI, dev_wO, dev_h1, dev_pred);
		int size_X = hidden_matrix_size.WB * hidden_matrix_size.HB;
		int mem_size_X = sizeof(float) * size_X;
		int size_wI = hidden_matrix_size.WA * hidden_matrix_size.HA;
		int mem_size_wI = sizeof(float) * size_wI;
		int size_wO = output_matrix_size.WA * output_matrix_size.HA;
		int mem_size_wO = sizeof(float) * size_wO;
		int size_h1 = hidden_matrix_size.WC * hidden_matrix_size.HC;
		int mem_size_h1 = sizeof(float) * size_h1;
		int size_pred = output_matrix_size.WC * output_matrix_size.HC;
		int mem_size_pred = sizeof(float) * size_pred;

		hipMalloc((void **)&dev_X, mem_size_X);
		checkCUDAError("hipMalloc dev_X");
		hipMalloc((void **)&dev_wI, mem_size_wI);
		checkCUDAError("hipMalloc dev_wI");
		hipMalloc((void **)&dev_wO, mem_size_wO);
		checkCUDAError("hipMalloc dev_wO");
		hipMalloc((void **)&dev_h1, mem_size_h1);
		checkCUDAError("hipMalloc dev_h1");
		hipMalloc((void **)&dev_pred, mem_size_pred);
		checkCUDAError("hipMalloc dev_pred");

		hipMemcpy(dev_X, Xi, mem_size_X, hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy dev_X");
		hipMemcpy(dev_wI, wI, mem_size_wI, hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy dev_wI");
		hipMemcpy(dev_wO, wO, mem_size_wO, hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy dev_wO");
		// allocate host memory for result
		//unsigned int size_pred = output_matrix_size.WC * output_matrix_size.HC;
		//unsigned int mem_size_pred = sizeof(float) * size_pred;
		float *h1 = (float *)malloc(mem_size_h1);

		hipblasHandle_t handle;
		hipblasCreate(&handle);

		//hidden layer
		dim3 threads(blockSize);
		dim3 grid((hidden_matrix_size.WC*hidden_matrix_size.HC + blockSize - 1) / blockSize);

		matrixMultiply(&handle, hidden_matrix_size, dev_wI, dev_X, dev_h1);
		hipMemcpy(h1, dev_h1, mem_size_pred, hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy pred");
		printf("\n\n Matriz h1:");
		printf("\n %f %f", h1[0], h1[1]);

		kernSigmoid <<<grid, threads>> > (hidden_matrix_size.HC*hidden_matrix_size.WC, dev_h1);
		checkCUDAError("kernSigmoid");


		//dim3 grid1(output_matrix_size.WC / threads.x, output_matrix_size.HC / threads.y);
		dim3 grid1((output_matrix_size.WC*output_matrix_size.HC + blockSize - 1) / blockSize);
		//output layer
		matrixMultiply(&handle, output_matrix_size, dev_wO, dev_h1, dev_pred);
		hipMemcpy(pred, dev_pred, mem_size_pred, hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy pred");
		printf("\n\n Matriz pred:");
		printf("\n %f", pred[0]);
		printf("\n");

		kernSigmoid2 << <grid1, threads >> > (output_matrix_size.HC*output_matrix_size.WC, dev_pred);
		checkCUDAError("kernSigmoid");

		hipMemcpy(pred, dev_pred, mem_size_pred, hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy pred");

		hipblasDestroy(handle);
		checkCUDAError("handle");

		//deviceMemory();
		hipFree(dev_X);
		hipFree(dev_wI);
		hipFree(dev_wO);
		hipFree(dev_h1);
		hipFree(dev_pred);
	}

	void train(float *X, float *y, int sizeData, const int hiddenNodes, const int numLabels, const int numData) {
		sMatrixSize hidden_matrix_size = {hiddenNodes, sizeData, 1, sizeData, 1, hiddenNodes };
		sMatrixSize output_matrix_size = {numLabels, hiddenNodes, 1, hiddenNodes, 1, numLabels };

		unsigned int size_wI = hidden_matrix_size.WA * hidden_matrix_size.WA;
		unsigned int mem_size_wI = sizeof(float) * size_wI;
		float *wI = (float *)malloc(mem_size_wI);

		unsigned int size_wO = output_matrix_size.HA * output_matrix_size.WA;
		unsigned int mem_size_wO = sizeof(float) * size_wO;
		float *wO = (float *)malloc(mem_size_wO);

		fixedInit(wI, size_wI);
		fixedInit(wO, size_wO);

		float *permuteData = (float *)malloc(numData);
		float *Xi = (float *)malloc(sizeData);
		float *yi = (float *)malloc(numLabels);

		unsigned int size_pred = output_matrix_size.WC * output_matrix_size.HC;
		unsigned int mem_size_pred = sizeof(float) * size_pred;
		float *pred = (float *)malloc(mem_size_pred);

		for (int iter = 0; iter < 1; iter++) {
			generateRandom(numData, permuteData);
			printf("predicting iteration %i \n", iter);
			for (int i = 0; i < numData; i++) {
				int index = permuteData[i];
				memcpy(Xi, (void **)&X[sizeData*index], sizeData * sizeof(float));
				memcpy(yi, (void **)&y[numLabels*index], numLabels * sizeof(float));

				printf("index %i \n", index);
				printf("data: %f %f label: %f \n", Xi[0] , Xi[1], yi[0]);

				forward(pred, Xi, wI, wO, hidden_matrix_size, output_matrix_size);
				for (int j = 0; j < numLabels; j++) {
					printf("prediction: %f \n", pred[j]);
				}
			}
			printf("forward done \n");
		}
		printf("predictions done \n");

		//free(wI);
		//free(wO);
		//free(Xi);
		//free(yi);
		//free(permuteData);
		//free(pred);
	}

	void testMatrixMultiply() {
		sMatrixSize matrix_size = { 3, 4, 3, 2, 2, 4};

		// allocate host memory for matrices A and B
		unsigned int size_A = matrix_size.WA * matrix_size.HA;
		unsigned int mem_size_A = sizeof(float) * size_A;
		float *h_A = (float *)malloc(mem_size_A);
		unsigned int size_B = matrix_size.WB * matrix_size.HB;
		unsigned int mem_size_B = sizeof(float) * size_B;
		float *h_B = (float *)malloc(mem_size_B);

		// set seed for rand()
		srand(2006);

		// initialize host memory
		indexInit(h_A, size_A);
		indexInit(h_B, size_B);

		// allocate device memory
		float *d_A, *d_B, *d_C;
		unsigned int size_C = matrix_size.WC * matrix_size.HC;
		unsigned int mem_size_C = sizeof(float) * size_C;

		// allocate host memory for the result
		float *h_C = (float *)malloc(mem_size_C);

		hipMalloc((void **)&d_A, mem_size_A);
		hipMalloc((void **)&d_B, mem_size_B);
		hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
		hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
		hipMalloc((void **)&d_C, mem_size_C);

		// setup execution parameters
		dim3 threads(blockSize, blockSize);
		dim3 grid(matrix_size.HB / threads.x, matrix_size.WA / threads.y);

		// create and start timer
		printf("Computing result using CUBLAS... \n");

		hipblasHandle_t handle;
		hipblasCreate(&handle);

		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		//matrixMultiply(&handle, matrix_size, d_A, d_B, d_C);
		MatrixMulCUDA <blockSize><< < grid, threads >> > (d_C, d_A, d_B, matrix_size.HA, matrix_size.HB);
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		// copy result from device to host
		hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

		// Destroy the handle
		hipblasDestroy(handle);

		printf("\n\n Matriz A:");
		printMat(h_A, matrix_size.WA, matrix_size.HA);
		printf("\n\n Matriz B:");
		printMat(h_B, matrix_size.WB, matrix_size.HB);
		printf("\n\n Matriz C:");
		printMat(h_C, matrix_size.WC, matrix_size.HC);
		printf("\n\n");

		// clean up memory
		free(h_A);
		free(h_B);
		free(h_C);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
	}

}
