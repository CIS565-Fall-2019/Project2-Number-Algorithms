#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_math_constants.h>
#include "common.h"
#include "mlp.h"

#define ALLOWKERNEL5 0
#define RANDSEED 0x0bad1bad2bad123
#define LAMBDA 0.1 //the learning delta

//These are definitions for index math in the 1d-2d world
#define UL(idx, w) (idx - w - 1)
#define UC(idx, w) (idx - w)
#define UR(idx, w) (idx - w + 1)
#define CL(idx, w) (idx - 1)
#define CC(idx, w) (idx)
#define CR(idx, w) (idx + 1)
#define DL(idx, w) (idx + w - 1)
#define DC(idx, w) (idx + w)
#define DR(idx, w) (idx + w + 1)


namespace CharacterRecognition {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	//##################################
	// SIZE DEFINES
	//##################################

#define NUMFILTERS 6
#define KERNWIDTH 3
#define POOLWIDTH 3

#define F0SIZE 10201

#define SINCONVRAWSIZE ((sqrt(F0SIZE) - (KERNWIDTH - 1)) * (sqrt(F0SIZE) - (KERNWIDTH - 1)))
#define CONVRAWSIZE (SINCONVRAWSIZE * NUMFILTERS)
#define SINCONVPOOLSIZE (CONVRAWSIZE / (POOLWIDTH * POOLWIDTH))
#define CONVPOOLSIZE (SINCONVPOOLSIZE * NUMFILTERS)
#define F1SIZE (CONVPOOLSIZE)

#define W1SIZE (F0SIZE * F1SIZE)

#define F2SIZE 500
#ifndef RSIZE
#define RSIZE 52
#endif
#define W2SIZE (F2SIZE * RSIZE)



	//##################################
	// DEVICE POINTER MEMORY
	//##################################

	float* dF0;//features 0 (orig data)
	float* dC0;//convolutional memory for first layer
	float* dF1;//features 1
	float* dW1;//weights 1
	float* dW1D;//delta value for weights 1
	float* dPj;//psi_j result matrix
	float* dOj;//omega_j result matrix
	float* dF2;//features 2
	float* dF2A;//features 2 (activated)
	float* dW2;//weights 2
	float* dW2D;//delta value for weights 2
	float* dPi;//psi_i result matrix
	float* dR;//result
	float* dRA;//result(activated)
	float* dRE;//result error



	//CONVOLUTIONAL MEMORY
	float* dCR;//convolutional output, raw
	float* dCP;//convolutional output, pooled

	//Convolution kernel initialization
	filter3 kern1 = {	1.0 / 16,	1.0 / 8,	1.0 / 16,
						1.0 / 8,	1.0 / 4,	1.0 / 8,
						1.0 / 16,	1.0 / 8,	1.0 / 16 };//gaussian
	filter3 kern2 = { -1, -1, -1, -1, 8, -1, -1, -1, -1 };//outline
	filter3 kern3 = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };//sobel top
	filter3 kern4 = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };//sobel right
	filter3 kern5 = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };//sobel bottom
	filter3 kern6 = { 1, 0, -1, 2, 0, -2, 1, 0, -1 };//sobel left
	filter3 allKernels[NUMFILTERS] = { kern1, kern2, kern3, kern4, kern5, kern6 };

	//##################################
	// FUNCTION DELCARATIONS
	//##################################

	/**
	Gets the "index" for the thread
	Currently, only supporting single-dimensional block indexes
	Computes all relevant x, y, z transformations
	*/
	__device__ int getIndex();

	//##################################
	// DEVICE POINTER MALLOC AND FREE
	//##################################

	void kmallocBuffers() {
		hipMalloc((void**)& dF0, F0SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dC0, CONVRAWSIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dF1, F1SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dW1, W1SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dW1D, W1SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dPj, F2SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dOj, F2SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dF2, F2SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dF2A, F2SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dW2, W2SIZE *sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dW2D, W2SIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dPi, RSIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dR, RSIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dRE, RSIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);
		hipMalloc((void**)& dRA, RSIZE * sizeof(float));
		checkCUDAErrorFn("hipMalloc failed\n", NULL, __LINE__);

	}//kmallocBuffers

	void kfreeBuffers() {
		hipFree(dF0);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dC0);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dF1);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dW1);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dW1D);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dPj);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dOj);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dF2);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dF2A);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dW2);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dW2D);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dPi);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dR);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dRE);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);
		hipFree(dRA);
		checkCUDAErrorFn("hipFree failed\n", NULL, __LINE__);

	}//kfreeBuffers
        
	//##################################
	// DEVICE FUNCTIONS
	//##################################

	__device__ int getIndex() {
		int threadIndex = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
		int overallIndex = threadIndex + blockIdx.x * (blockDim.x * blockDim.y * blockDim.z);

		return overallIndex;
	}//getIndex

	//##################################
	// DEVICE GLOBAL FUNCTIONS
	//##################################

	__global__ void kTranspose(float* A, float* Aswap, int m, int n) {
		int index = getIndex();
		if (index >= m * n) return;

		int srcR = index / n;
		int srcC = index % n;
		int dstR = srcC;
		int dstC = srcR;
		//int srcIndex = srcR * n + srcC;
		//int dstIndex = dstR * m + dstC;
		//Aswap[dstIndex] = A[srcIndex];
		Aswap[dstR * m + dstC] = A[srcR * n + srcC];
	}//kTranspose

	/**
	Performs our activation function on our results to put them in the range between 0 and 1
	Does so in-place
	*/
	__global__ void kActivateResults(float* results, float* resultsA, int N) {
		int index = getIndex();
		if (index >= N) return;
		resultsA[index] = 1.0 / (1.0 + expf(-1 * results[index]));
	}//activateResults

	__global__ void kActivateInverse(float* results, float* resultsIA, int N) {
		int index = getIndex();
		if (index >= N) return;
		//resultsIA[index] = logf(results[index] / (1.0 - results[index]));
		float ex = expf(results[index]);
		resultsIA[index] = ex / ((ex + 1) * (ex + 1));
	}//kActivateInverse




	//##################################
	// HOST HELPER FUNCTIONS
	//##################################

	void activateResults(float* results, float* resultsActivated, int numResults) {
		dim3 tpb = dim3(BLOCKSIZE);
		dim3 bpg = dim3((numResults + BLOCKSIZE - 1) / BLOCKSIZE);

		kActivateResults<<<bpg, tpb>>>(results, resultsActivated, numResults);
		checkCUDAErrorFn("kActivateResults failed\n", NULL, __LINE__);
	}//activateResults

	__global__ void shiftByFactor(float* A, int N, float mulFactor, float offset) {
		int index = getIndex();
		if (index > N) return;
		A[index] = mulFactor * A[index] + offset;
	}//shiftByFactor

	void gpuFillRand(float* A, int nr_rows_A, int nr_cols_A, float lo, float hi){
		hiprandGenerator_t prng;
		hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);

		hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) RANDSEED);

		hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);

		//shift the random numbers into the given range
		float mulFactor = hi - lo;
		float offset = lo;
		
		int numElements = nr_rows_A * nr_cols_A;
		dim3 tpb = dim3(BLOCKSIZE);
		dim3 bpg = dim3((numElements + BLOCKSIZE - 1) / BLOCKSIZE);

		shiftByFactor<<<bpg, tpb>>>(A, numElements, mulFactor, offset);
		checkCUDAErrorFn("shiftByFactor failed\n", NULL, __LINE__);
		hipDeviceSynchronize();//safety
	}//gpuFillRand



	void matMul(hipblasHandle_t* handle, const float* A, const float* B, float* C, int m, int k, int n) {

		//Since cublas expects column-major indexing, our A is effectively AT (kxm), and our B is effectively BT (nxk)
		//As such, we're going to be doing BT * AT = CT (nxm)
		//Then, we transpose C "in place" before we return
		//And by that I mean we don't do that, because for some reason the multiplication works how I want
		float alpha = 1.0;
		float beta = 0.0;

		//Future development: put the result into Cswap, transpose into C
		hipblasSgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, B, n, A, k, &beta, C, n) ;
		checkCUDAErrorFn("the internal matrix multiply failed\n", NULL, __LINE__);
		//hipDeviceSynchronize();

		//no need to transpose?? not sure why, but this function operates
		//transpose(C, Cswap, n, m);
	}//matMul

	//##################################
	// ERROR CALCULATIONS (?)
	//##################################

	float_v calcErrorSingle(InputData record, float* resultArray, float* kResultArray) {
		float_v retval = float_v();
		float_v trueResult = record.resultArray;
		for (int i = 0; i < trueResult.size(); i++) {
			float error = resultArray[i] - trueResult[i];
			retval.push_back(error);
		}//for

		if (kResultArray) {
			hipMemcpy(kResultArray, retval.data(), trueResult.size() * sizeof(float), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);
		}//if

		return retval;
	}//calcError

	float_v calcSumSquareErrors(float_vv errorVals) {
		float_v result = float_v(errorVals[0].size(), 0.0f);
		for (int i = 0; i < errorVals.size(); i++) {
			for (int j = 0; j < errorVals[0].size(); j++) {
				result[j] += errorVals[i][j] * errorVals[i][j] / 2.0;
			}//for j
		}//for i

		return result;
	}//calcSumSquareErrors

	float calcEnergy(float_v errors) {
		float sum = 0;
		for (int i = 0; i < errors.size(); i++) {
			sum += (errors[i] * errors[i]);
		}//for
		return sum / errors.size();//averaging the energy function?
	}//calcEnergy

	//##################################
	// WEIGHT CHANGES
	//##################################

	__global__ void kCalcWeightChange1(float* thetaA, float* omega, float* data, int cmax, int rmax,
		float* weightChange, float* psiOut) {
		int index = getIndex();
		if (index >= rmax * cmax) return;

		int r = index / cmax;
		int c = index % cmax;

		float rA = thetaA[r];
		float psi = (rA * (1 - rA)) * omega[r];
		weightChange[index] = -1.0 * LAMBDA * psi * data[c];
		psiOut[r] = psi;
		return;

	}//kCalcWeightChange1

	__global__ void kCalcWeightChange2(float* thetaA, float* omegaError, float* data, int cmax, int rmax, 
					float* weightChange, float* psiOut) {
		int index = getIndex();
		if (index >= rmax * cmax) return;

		int r = index / cmax;
		int c = index % cmax;

		float rA = thetaA[r];
		float psi = (rA * (1 - rA)) * omegaError[r];
		weightChange[index] = -1.0 * LAMBDA * psi * data[c];
		psiOut[r] = psi;
		return;

	}//kCalcWeightChange2

	void calcWeightChange1(float* thetaResultA, float* omegaError, float* features, int kmax, int jmax, float* weightChange, float* psiOut) {
		dim3 tpb = dim3(BLOCKSIZE);
		dim3 bpgij = dim3(((jmax * kmax) + BLOCKSIZE - 1) / BLOCKSIZE);

		kCalcWeightChange1<<<bpgij, tpb>>>(thetaResultA, omegaError, features, kmax, jmax, weightChange, psiOut);

	}//calcWeightChange1

	void calcWeightChange2(float* thetaResultA, float* omegaError, float* features, int jmax, int imax, float* weightChange, float* psiOut) {
		/*
		result: [0:imax)(52), error: [0:imax)(52), data: [0, jmax)(10201), weightChange (outvar) ixj matrix
		*/
		dim3 tpb = dim3(BLOCKSIZE);
		dim3 bpgij = dim3(((imax * jmax) + BLOCKSIZE - 1) / BLOCKSIZE);

		kCalcWeightChange2<<<bpgij, tpb>>>(thetaResultA, omegaError, features, jmax, imax, weightChange, psiOut);

	}//calcWeightChange

	//##################################
	// HOST MAIN FUNCTIONS
	//##################################

	void backPropagate(hipblasHandle_t* handle) {
		/*
		Derivative of E wrt weights at (j,i) (up to 10201 and 52, respectively):
			-sum of sse error * invActivate(result)[i] * hiddenData[j]
		psi_i = error_i * invActivate(result[i])
		Theta_i = result[i]
		omega_i = error_i
		Psi_j	= Omega_j * invActivate(Theta_j)
		Theta_j = hiddenData[j]
		Omega_j = (sum(psi_i) for i)

		weightChange_ji = lambda * hiddenData[j] * psi_i
		weightChange_kj = lambda * hiddenData[k] * psi_i

		*/
		hipblasHandle_t mHandle; bool handling = false;
		if (handle == NULL) {
			handling = true;
			handle = &mHandle;
			hipblasCreate(handle);
		}//if
		float alpha = 1.0;

		float testOut[F2SIZE];

		//final layer weight delta calculation
		calcWeightChange2(dRA, dRE, dF2A, F2SIZE, RSIZE, dW2D, dPi);
		//apply the weight change
		hipblasSaxpy(*handle, F2SIZE * RSIZE, &alpha, dW2D, 1, dW2, 1);

		//calculate Omega_j off the psi_i values
		matMul(handle, dW2, dPi, dOj, F2SIZE, RSIZE, 1);
		checkCUDAErrorFn("matMul failed\n", NULL, __LINE__);

		//DEBUG OUTPUT
		hipMemcpy(testOut, dPi, RSIZE * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("cudamemcpy failed\n", NULL, __LINE__);

		//next-to-last layer weight delta calculation
		calcWeightChange1(dF2A, dOj, dF0, F0SIZE, F2SIZE, dW1D, dPj);
		checkCUDAErrorFn("calcWeightChange failed\n", NULL, __LINE__);
		//apply the weight change
		hipblasSaxpy(*handle, F0SIZE * F2SIZE, &alpha, dW1D, 1, dW1, 1);
		checkCUDAErrorFn("saxpy failed\n", NULL, __LINE__);

		hipMemcpy(testOut, dW1, F2SIZE * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("cudamemcpy failed\n", NULL, __LINE__);


		if (handling) {
			hipblasDestroy(*handle);
		}//if
	}//backPropagate

	float_v forwardPropagate(InputData x, float* resultArray, hipblasHandle_t* handle) {
		//Make our cublas handle if not handed one
		hipblasHandle_t mHandle; bool handling = false;
		if (handle == NULL) {
			handling = true;
			handle = &mHandle;
			hipblasCreate(handle);
		}//if
		float* dataPtr = x.fData.data();
		float testOut[F2SIZE] = {};

		hipMemcpy(dF0, dataPtr, F0SIZE * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);

		//Fully connected layer w/ W1
		matMul(handle, dF0, dW1, dF2, 1, F0SIZE, F2SIZE);
		checkCUDAErrorFn("matMul failed\n", NULL, __LINE__);

		//DEBUG OUTPUT
		hipMemcpy(testOut, dW1, F2SIZE * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);
		
		//activate the first results
		activateResults(dF2, dF2A, F2SIZE);
		checkCUDAErrorFn("activateResults failed\n", NULL, __LINE__);

		//Fully connected layer w/ W2
		matMul(handle, dF2A, dW2, dR, 1, F2SIZE, RSIZE);
		checkCUDAErrorFn("matMul failed\n", NULL, __LINE__);

		//Activate results
		activateResults(dR, dRA, RSIZE);
		checkCUDAErrorFn("activateResults failed\n", NULL, __LINE__);

		hipMemcpy(resultArray, dRA, RSIZE * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);

		if (handling) {
			hipblasDestroy(*handle);
		}//if

		return calcErrorSingle(x, resultArray, dRE);
	}//forwardPropH

	void trainWeights(InputData_v records, int numIterations) {
		hipblasHandle_t handle;
		hipblasCreate(&handle);

		float results[RSIZE] = {};//floating space for the results to be put

		//initialize random weights between -1 and 1
		gpuFillRand(dW1, F1SIZE, F2SIZE, -1.0, 1.0);
		gpuFillRand(dW2, F2SIZE, RSIZE, -1.0, 1.0);

		for (int iter = 0; iter < numIterations; iter++) {
			float_vv errorValues = float_vv();

			//go forward
			float_v errorVal = forwardPropagate(records[iter % records.size()], results, &handle);

			/*
			errorValues.push_back(errorVal);
			float_v sseError = calcSumSquareErrors(errorValues);
			*/
			if (iter == 0) {
				printf("==========RESULTS=========\n");
				for (int i = 0; i < RSIZE; i++) {
					printf("@%02d:  %f\t", i, results[i]);
					if ((i + 1) % 4 == 0) {
						printf("\n");
					}
				}//for
			}//if
			float energy = calcEnergy(errorVal);
			printf("@%03d: Calculated energy is %.8f\n", iter, energy);

			//go backwards
			backPropagate(&handle);
		}//for

		hipblasDestroy(handle);

	}//trainWeights


	void transpose(float* A, float* Aswap, int m, int n) {
		int numElements = m * n;
		dim3 tpb = dim3(BLOCKSIZE);
		dim3 bpg = dim3((numElements + BLOCKSIZE - 1) / BLOCKSIZE);

		float testArray[2][2];

		hipMemcpy(testArray, A, 4 * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);

		kTranspose << <tpb, bpg >> > (A, Aswap, m, n);
		checkCUDAErrorFn("kTranspose failed\n", NULL, __LINE__);

		hipMemcpy(testArray, Aswap, 4 * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);

		hipMemcpy(A, Aswap, numElements * sizeof(float), hipMemcpyDeviceToDevice);
		checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);

		hipMemcpy(testArray, A, 4 * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAErrorFn("hipMemcpy failed\n", NULL, __LINE__);

		return;

	}//transpose


	//##################################
	// CONVOLVING
	//##################################

	//Convolutional layer:
	//1. Convolve (into an intermediary)
	//2. Activate the intermediary
	//3. Max pool down into some feature vector (to be fed into some of the FC layers)

	/**
	Pools some number of activated convolutions down into a smaller buffer
	Does so in blockWidth x blockWidth squares
	Wants to spawn a number of threads equal to the number of resultant output "pixels"
	*/
	__global__ void kmaxPool(float* idata, float* odata, int blockWidth, int idataWidth, int odataWidth) {
		int index = getIndex();
		if (index >= odataWidth * odataWidth) return;

		int oR	= index / odataWidth;
		int oC	= index % odataWidth;
		int iR	= oR * blockWidth - (blockWidth / 2);
		int iC	= oC * blockWidth - (blockWidth / 2);
		int iindex = iR * idataWidth + iC;
		float max = -1.0e40;//stand-in for a minimum
		for (int i = 0; i < blockWidth; i++) {
			int iOffset = idataWidth * (i - (blockWidth / 2));
			for (int j = 0; j < blockWidth; j++) {
				max = fmaxf(max, idata[iindex + iOffset + (j - (blockWidth / 2))]);
			}//for
		}//for
		odata[index] = max;
	}//kmaxPool
		
	/**
	* Does a convolution from one image to another
	* A few notes:
	* Takes char data in for the input
	* Assuming we're running one thread per output pixel, and that we've sized things correctly for our filter
	* filter, idata, and odata must all be square
	* Also, currently only accepting filter widths of 3
	*/
	__global__ void kconvolve(filter3 filter, float* idata, float* odata, int odataWidth) {
		int index = getIndex();
		if (index >= odataWidth * odataWidth) return;
		int idataW = odataWidth + 2;

		//get ourselves an "idata" index
		int iindex = (index / odataWidth) * 2 + 1 + idataW;

		float sum = 0;

		uint8_t relData[9];
		//Flips the kernel here
		relData[0] = idata[DR(iindex, idataW)];
		relData[1] = idata[DC(iindex, idataW)];
		relData[2] = idata[DL(iindex, idataW)];
		relData[3] = idata[CR(iindex, idataW)];
		relData[4] = idata[CC(iindex, idataW)];
		relData[5] = idata[CL(iindex, idataW)];
		relData[6] = idata[UR(iindex, idataW)];
		relData[7] = idata[UC(iindex, idataW)];
		relData[8] = idata[UL(iindex, idataW)];
		for (int i = 0; i < 9; i++) {
			sum += relData[i] * filter.kernel[i];
		}//for 9
#if ALLOWKERNEL5
		else if (filterWidth == 5) {
			uint8_t relData[25];
			//Flips the kernel here (without the macro stuff)
			for (int i = 0; i < 5; i++) {
				int iOffset = idataW * (i - 2);
				for (int j = 0; j < 5; j++) {
					relData[5 * i + j] = idata[iindex + (j - 2) + iOffset];
				}//for
			}//for
			for (int i = 0; i < 25; i++) {
				sum += relData[i] * filter[i];
			}//for 25
		}//elif 5
#endif

		odata[index] = sum;

	}//kconvolve

	void convolve(float* idata, float* odata, int odataSize, filter3 kernel) {
		dim3 tpb = dim3(BLOCKSIZE);
		dim3 bpg = dim3(((odataSize) + BLOCKSIZE - 1) / BLOCKSIZE);

		kconvolve<<<bpg, tpb>>>(kernel, idata, odata, (int)sqrt(odataSize));
		checkCUDAErrorFn("kconvolve failed\n", NULL, __LINE__);
	}//convolve

	/**
	Does the forward propagation for convolving stuff
	Also max-pools
	Returns the size of the output layer (sure why not)
	*/
	int convolveStep(float* inputLayer, int inputLayerSize, float* outputPoolingLayer, float* outputLayer, int poolWidth) {
		int inputLayerWidth = (int)sqrt(inputLayerSize);
		int outputPoolingBlockWidth = inputLayerWidth - 2;
		int outputPoolingBlockSize = outputPoolingBlockWidth * outputPoolingBlockWidth;
		int outputPooledBlockSize = outputPoolingBlockSize / (poolWidth * poolWidth);
		int outputPooledBlockWidth = (int)sqrt(outputPooledBlockSize);
		int outputLayerSize = NUMFILTERS * outputPooledBlockSize;

		//convolve
		for (int i = 0; i < NUMFILTERS; i++) {
			convolve(inputLayer, outputPoolingLayer + (i * outputPoolingBlockSize), outputPoolingBlockSize, allKernels[i]);
		}//for

		hipDeviceSynchronize();

		//pool
		dim3 tpb = dim3(BLOCKSIZE);
		dim3 bpg = dim3(((outputPooledBlockSize)+BLOCKSIZE - 1) / BLOCKSIZE);
		for (int i = 0; i < NUMFILTERS; i++) {
			//	__global__ void kmaxPool(float* idata, float* odata, int blockWidth, int idataWidth, int odataWidth) {
			int iBlockOffset = i * outputPoolingBlockSize;
			int oBlockOffset = i * outputPooledBlockSize;
			kmaxPool<<<bpg, tpb>>>(outputPoolingLayer + iBlockOffset, outputLayer + oBlockOffset, poolWidth, outputPoolingBlockWidth, outputPooledBlockWidth);
			checkCUDAErrorFn("kmaxpool failed\n", NULL, __LINE__);
		}//for

		return outputLayerSize;
	}//convolveStep

}//CharacterRecognition
