#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

			thrust::device_vector<int> in(idata, idata+n);
			thrust::device_vector<int> out(n);

            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            thrust::exclusive_scan(in.begin(),in.end(),out.begin());
			thrust::copy(out.begin(), out.end(), odata);
            timer().endGpuTimer();
        }
    }
}
