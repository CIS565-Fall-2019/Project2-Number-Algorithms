#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			thrust::host_vector<int> dv_in(idata, idata + n);
			thrust::host_vector<int> dv_out(odata, odata + n);

			thrust::device_vector<int> dev_in(dv_in);
			thrust::device_vector<int> dev_out(dv_out);

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            thrust::exclusive_scan(dev_in.begin(), dev_in.end(), dev_out.begin());
            timer().endGpuTimer();
			thrust::copy(dev_out.begin(), dev_out.end(), dv_out.begin());
			hipMemcpy(odata, &dv_out[0], sizeof(int) * n, hipMemcpyHostToHost);
			checkCUDAError("memcopy error in thryust scan");
        }
    }
}
