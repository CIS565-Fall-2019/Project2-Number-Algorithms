#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int* dev_idata;
			int* dev_odata;
			hipMalloc((void**)&dev_idata, sizeof(int) * n);
			hipMalloc((void**)&dev_odata, sizeof(int) * n);

			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			thrust::device_ptr<int> dv_in(dev_idata);
			thrust::device_ptr<int> dv_out(dev_odata);

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
			thrust::exclusive_scan(dv_in, dv_in + n, dv_out);
            timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);

			hipFree(dev_idata);
			hipFree(dev_odata);
        }
    }
}
