#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int * dev_idata;
			int * dev_odata;
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_idata failed!");
			thrust::device_ptr<int> input(dev_idata);
			thrust::device_ptr<int> output(dev_odata);

			timer().startGpuTimer();
			thrust::exclusive_scan(input, input + n, output);
			timer().endGpuTimer();
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_idata);
			hipFree(dev_odata);
        }
    }
}
