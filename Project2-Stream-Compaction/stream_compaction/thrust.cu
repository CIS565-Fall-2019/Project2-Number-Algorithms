#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int *dev_in;
			hipMalloc((void**)&dev_in, n * sizeof(int));
			checkCUDAError("hipMalloc dev_in failed!");

			// copy input to device buffer
			hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_in idata failed!");

			thrust::device_vector<int> dev_thrust_in(dev_in, dev_in + n);
			thrust::device_vector<int> dev_thrust_out(n);

            timer().startGpuTimer();
            thrust::exclusive_scan(dev_thrust_in.begin(), dev_thrust_in.end(), dev_thrust_out.begin());
			checkCUDAError("thrust::exclusive_scan failed!");
            timer().endGpuTimer();

			int *dev_out = thrust::raw_pointer_cast(dev_thrust_out.data());
			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy odata dev_out failed!");

			hipFree(dev_in);
			checkCUDAError("hipFree dev_in failed!");
        }
    }
}
