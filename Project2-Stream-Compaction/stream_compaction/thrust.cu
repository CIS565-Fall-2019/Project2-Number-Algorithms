#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg)

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

			int *dev_idata, *dev_odata;
			hipMalloc((void **)&dev_odata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_odata failed!");
			hipMalloc((void **)&dev_idata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			thrust::device_ptr<int> dev_idataItr(dev_idata);
			thrust::device_ptr<int> dev_odataItr(dev_odata);

			thrust::exclusive_scan(dev_idataItr, dev_idataItr + n, dev_odataItr);

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);


            timer().endGpuTimer();
        }
    }
}
