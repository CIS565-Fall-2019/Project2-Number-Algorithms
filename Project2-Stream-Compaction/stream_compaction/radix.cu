#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "common.h"
#include "radix.h"
#include "efficient.h"
#include ""
#include <math.h>

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
#define MAX_BIT 6
#define blockSize 128

int* data;
int* output;

int* B;
int* E;
int* F;
int* T;
int* D;

namespace StreamCompaction {
	namespace Radix {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}


		__global__ void compute_b_e(int n, int position, int* input, int* B, int* E) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}
			int value = input[index];
			int bit = (value >> position) & 1;
			if (bit == 0) {
				B[index] = 0;
				E[index] = 1;
			}
			else {
				B[index] = 1;
				E[index] = 0;
			}
		}

		__global__ void compute_total_falses(int n, int* totalFalses, int* E, int* F) {
			*totalFalses = E[n - 1] + F[n - 1];
		}

		__global__ void compute_t(int n, int* F, int* totalFalses, int* T) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}
			T[index] = index - F[index] + (*totalFalses);
		}

		__global__ void compute_d(int n, int* B, int* T, int* F, int* D) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}
			D[index] = B[index] ? T[index] : F[index];
		}

		__global__ void scatter(int n, int* indices, int* inp, int* op) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}
			op[indices[index]] = inp[index];
		}
		
		void sort(int n, int* odata, int*idata) {
			hipMalloc((void**)&data, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc data failed!");

			hipMalloc((void**)&output, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc output failed!");

			hipMemcpy(data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			hipMalloc((void**)&B, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc B failed!");

			hipMalloc((void**)&E, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc E failed!");

			hipMalloc((void**)&F, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc F failed!");

			hipMalloc((void**)&T, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc T failed!");

			hipMalloc((void**)&D, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc D failed!");

			int* totalFalses;
			hipMalloc((void**)&totalFalses, sizeof(int));
			checkCUDAErrorWithLine("hipMalloc totalFalses failed!");
			
			timer().startGpuTimer();
			
			for (int i = 0; i < MAX_BIT; i++) {
				dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
				compute_b_e << <fullBlocksPerGrid, blockSize >>> (n, i, data, B, E);

				//Scan E and store in F
				StreamCompaction::Efficient::scan_device(n, F, E, blockSize);

				compute_total_falses << <1,1>> > (n, totalFalses, E, F);

				compute_t << <fullBlocksPerGrid, blockSize >> > (n, F, totalFalses, T);

				compute_d << <fullBlocksPerGrid, blockSize >> > (n, B, T, F, D);

				//Scatter as per D
				scatter << <fullBlocksPerGrid, blockSize >> > (n, D, data, output);

				//Copy output back to input
				hipMemcpy(data, output, sizeof(int) * n, hipMemcpyDeviceToDevice);
			}

			timer().endGpuTimer();

			hipMemcpy(odata, output, sizeof(int) * n, hipMemcpyDeviceToHost);
		}
	}
}