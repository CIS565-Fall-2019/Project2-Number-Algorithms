#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
		int blockSize = 128;
		dim3 threadsPerBlock(blockSize);

		__global__ void kernSumPairs(int N, int d, int *srcArray, int *dstArray) {

		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int bufferLength = 1 << ilog2ceil(n);
			int *dev_arrayA;
			int *dev_arrayB;

			hipMalloc((void**)&dev_arrayA, bufferLength * sizeof(int));
			checkCUDAError("hipMalloc dev_arrayA failed!");
			
			hipMalloc((void**)&dev_arrayB, bufferLength * sizeof(int));
			checkCUDAError("hipMalloc dev_arrayB failed!");

			hipMemset(dev_arrayA, 0, bufferLength * sizeof(int));
			hipMemset(dev_arrayB, 0, bufferLength * sizeof(int));

			hipMemcpy(dev_arrayA, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_arrayB, idata, n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();

            // TODO
			int alternator = 0;
			for (int d = 0; d < ilog2ceil(n); ++d) {
				int numThreads = pow(2, d);
				dim3 blocksPerGrid((numThreads + blockSize - 1) / blockSize);

				if (alternator % 2 == 0) {
					kernSumPairs<<<blocksPerGrid, threadsPerBlock>>>(numThreads, d, dev_arrayA, dev_arrayB);
					hipMemcpy(dev_arrayA, dev_arrayB, bufferLength * sizeof(int), hipMemcpyDeviceToDevice);
					alternator++;

				}
				else {
					kernSumPairs<<<blocksPerGrid, threadsPerBlock>>>(numThreads, d, dev_arrayB, dev_arrayA);
					hipMemcpy(dev_arrayA, dev_arrayB, bufferLength * sizeof(int), hipMemcpyDeviceToDevice);
					alternator++;

				}

			}


            timer().endGpuTimer();


			hipFree(dev_arrayA);
			hipFree(dev_arrayB);
        }
    }
}
