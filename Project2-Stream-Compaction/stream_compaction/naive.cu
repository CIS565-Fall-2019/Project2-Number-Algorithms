#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			// Allocate buffers on GPU and move data in
			const size_t ARR_LEN = n * sizeof(int);
			const int NUM_THREADS = n;
			const int NUM_BLOCKS = 1;
			int* dev_odata;
			int* dev_tmp;

			hipMalloc(&dev_odata, ARR_LEN);
			hipMalloc(&dev_tmp,   ARR_LEN);

			// Copy input to odata buffer
			// After each loop of the algorithm we will swap tmp and odata
			// So that the final result will always be located in the dev_odata buffer.

			hipMemcpy(dev_odata, idata, ARR_LEN, ::hipMemcpyHostToDevice);
			hipMemcpy(dev_tmp,   idata, ARR_LEN, ::hipMemcpyHostToDevice);

			// Algorithm adapted from GPU Gems 3, Section 39.2.1
			/*
				1: for d = 1 to log2 n do
				2:   for all k in parallel do
				3:     if k >= 2^(d-1)  then
				4:     x[k] = x[k � 2^(d-1)] + x[k]
			*/
            timer().startGpuTimer();
			int* INSPECT_TMP = (int*)malloc(n * sizeof(int));
			int* INSPECT_ODATA = (int*)malloc(n * sizeof(int));
			for (int d = 1; d <= ilog2ceil(n); d++) {
				std::swap(dev_tmp, dev_odata);
#if _DEBUG
				hipMemcpy(INSPECT_ODATA, dev_odata, ARR_LEN, ::hipMemcpyDeviceToHost);
				hipMemcpy(INSPECT_TMP, dev_tmp, ARR_LEN, ::hipMemcpyDeviceToHost);
#endif
				kernScanStep<<<NUM_BLOCKS, NUM_THREADS >>>(n, d, dev_odata, dev_tmp);
#if _DEBUG
				hipMemcpy(INSPECT_ODATA, dev_odata, ARR_LEN, ::hipMemcpyDeviceToHost);
				hipMemcpy(INSPECT_TMP, dev_tmp, ARR_LEN, ::hipMemcpyDeviceToHost);
#endif
				hipDeviceSynchronize();
			}
			std::swap(dev_tmp, dev_odata);
			kernInclusiveToExclusive<<<NUM_BLOCKS, NUM_THREADS>>>(n, dev_odata, dev_tmp);
			hipMemset(dev_odata, 0, sizeof(int)); // Set first element to 0 (identity)
            timer().endGpuTimer();

			// Copy back to host and free memory
			hipMemcpy(odata, dev_odata, ARR_LEN, ::hipMemcpyDeviceToHost);

			hipFree(dev_tmp);
			hipFree(dev_odata);
        }

		__global__ void kernScanStep(const int N, const int D, int *out, const int* in) {
			int k = threadIdx.x + (blockIdx.x * blockDim.x);
			if (k >= N) {
				return;
			}

			if (k >= (int)powf(2, D - 1)) {
				out[k] = in[k - (int)powf(2, D - 1)] + in[k];
			}
			else {
				out[k] = in[k];
			}
		}

		__global__ void kernInclusiveToExclusive(const int N, int *out, const int* in) {
			int k = threadIdx.x + (blockIdx.x * blockDim.x);
			if (k >= N - 1) { // Modified condition, we do NOT want the last thread working.
				return;
			}

			out[k + 1] = in[k];
		}
    }
}
