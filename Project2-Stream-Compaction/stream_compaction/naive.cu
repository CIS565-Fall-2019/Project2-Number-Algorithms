#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include "common.h"
#include "naive.h"
#include <math.h>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		//####################
		//FUNCTION DEFINITIONS
		//####################
		
		__device__ void kShiftIncEx(int N, int index, int* idata, int* odata);
		__device__ void scanStep(int N, int index, unsigned long stepLevel, int* idata, int* odata);
		__device__ void kmoveData(int N, int index, int* odata, int* idata);
		__global__ void kScan(int N, int* idata, int* odata, int numLevels);

		int* kern_idata;
		int* kern_odata;

		__device__ void scanStep(int N, int index, unsigned long stepLevel, int* idata, int* odata) {
			if (index >= N) return;
			if (index < stepLevel) {
				odata[index] = idata[index];
				return;
			}//if low index
			odata[index] = idata[index] + idata[index - stepLevel];
			return;
		}//scanStep

		__device__ void kmoveData(int N, int index, int* odata, int* idata) {
			if (index >= N) return;
			int temp = odata[index];
			idata[index] = temp;
			return;
		}//moveData

		/**
		* Note: the numLevels is that log_2(N), and should be computed GPU-side
		*/
		__global__ void kScan(int N, int* idata, int* odata, int numLevels) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			int currentLevel = 0;
			unsigned long stepLevel = 1;
			while (currentLevel < numLevels) {
				stepLevel = 1 << currentLevel;
				scanStep(N, index, stepLevel, idata, odata);
				__syncthreads();
				kmoveData(N, index, odata, idata);
				__syncthreads();
				currentLevel++;
			}//while

			kShiftIncEx(N, index, idata, odata);

		}//kScan

		/**
		* Shifts our inclusive scan over to be an exclusive scan
		*/
		__device__ void kShiftIncEx(int N, int index, int* idata, int* odata) {
			if (index >= N) return;
			else if (index == 0) odata[index] = 0;
			else {
				odata[index] = idata[index - 1];
			}
		}//kShiftIncEx

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int threadsPerBlock = (n + BLOCKSIZE - 1) / BLOCKSIZE;
			dim3 tpb = dim3(threadsPerBlock);
			dim3 bpg = dim3(BLOCKSIZE);


			//Allocate memory
			hipMalloc((void**)& kern_idata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc kern_idata failed!\n", NULL, __LINE__);
			hipMalloc((void**)& kern_odata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc kern_odata failed!\n", NULL, __LINE__);

			//copy input over
			hipMemcpy(kern_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy kern_idata failed!\n", NULL, __LINE__);

			timer().startGpuTimer();


			int numLevels = ilog2ceil(n);

			kScan<<<tpb, bpg>>>(n, kern_idata, kern_odata, numLevels);
			checkCUDAErrorFn("kScan failed!\n", NULL, __LINE__);

			timer().endGpuTimer();

			hipMemcpy(odata, kern_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpy kern_odata failed!\n", NULL, __LINE__);

			hipFree(kern_idata);
			hipFree(kern_odata);

        }
    }
}
