#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include ""
#include <math.h>

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define blockSize 128

int* dev_A;
int* dev_B;

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
		__global__ void updateSum(int N, int d, int* input, int* output) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) {
				return;
			}
			int offset = 1 << (d - 1);
			if (index >= offset) {
				output[index] = input[index - offset] + input[index];
			}
			else {
				output[index] = input[index];
			}
		}
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
			int iterations = ilog2ceil(n);

			hipMalloc((void**)&dev_A, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_A failed!");

			hipMalloc((void**)&dev_B, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_B failed!");

			hipMemcpy(dev_A, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			for (int d = 1; d <= iterations; d++) {
				dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
				updateSum << <fullBlocksPerGrid, blockSize >> > (n, d, dev_A, dev_B);
				std::swap(dev_A, dev_B);
				/*if (d % 2 == 0) {
					updateSum << <fullBlocksPerGrid, blockSize >> > (n, d, dev_A, dev_B);
				}
				else {
					updateSum << <fullBlocksPerGrid, blockSize >> > (n, d, dev_B, dev_A);
				}*/
			}

			/*if ((iterations) % 2 == 0) {
				hipMemcpy(odata, dev_B, sizeof(int) * n, hipMemcpyDeviceToHost);
			}
			else {
				hipMemcpy(odata, dev_A, sizeof(int) * n, hipMemcpyDeviceToHost);
			}*/
			hipMemcpy(odata+1, dev_A, sizeof(int) * (n-1), hipMemcpyDeviceToHost);
			odata[0] = 0;
            timer().endGpuTimer();
        }
    }
}
