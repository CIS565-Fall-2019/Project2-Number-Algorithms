#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include ""
#include <math.h>

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

//#define blockSize 128

int* dev_A;
int* dev_B;

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
		__global__ void updateSum(int N, int d, int* input, int* output) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) {
				return;
			}
			int offset = 1 << (d - 1);
			if (index >= offset) {
				output[index] = input[index - offset] + input[index];
			}
			else {
				output[index] = input[index];
			}
		}
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, int blockSize) {
			hipMalloc((void**)&dev_A, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_A failed!");

			hipMalloc((void**)&dev_B, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_B failed!");

			hipMemcpy(dev_A, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			timer().startGpuTimer();
			int iterations = ilog2ceil(n);
			for (int d = 1; d <= iterations; d++) {
				dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
				updateSum << <fullBlocksPerGrid, blockSize >> > (n, d, dev_A, dev_B);
				std::swap(dev_A, dev_B);
			}
            timer().endGpuTimer();
			hipMemcpy(odata + 1, dev_A, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
			odata[0] = 0;
        }
    }
}
