#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernInclusiveScanIteration(int n, int iteration, int *out, int *in) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			
			int nextIndex = ceil(pow(2.0, double(iteration - 1))); // encountered rounding issue at 2048, should fix it
			if (index >= nextIndex) {
				out[index] = in[index - nextIndex] + in[index];
			}
		}

		__global__ void kernShiftRight(int n, int *out, int *in) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}

			out[index] = index == 0 ? 0 : in[index - 1];
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int *dev_in;
			int *dev_out;

			// malloc device buffers
			hipMalloc((void**)&dev_in, n * sizeof(int));
			checkCUDAError("hipMalloc dev_in failed!");
			hipMalloc((void**)&dev_out, n * sizeof(int));
			checkCUDAError("hipMalloc dev_out failed!");

			// copy input to device
			hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_in idata failed!");
			hipMemcpy(dev_out, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_out idata failed!");

			// perform inclusive scan
			timer().startGpuTimer();

			dim3 gridSize = dim3((n + blockSize - 1) / blockSize, 1, 1);
			for (int d = 1; d <= ilog2ceil(n); d++) {
				// run one iteration
				kernInclusiveScanIteration<<<gridSize, blockSize>>>(n, d, dev_out, dev_in);
				checkCUDAError("kernInclusiveScanIteration failed!");

				// copy out to in
				hipMemcpy(dev_in, dev_out, n * sizeof(int), hipMemcpyDeviceToDevice);
				checkCUDAError("hipMemcpy dev_in dev_out failed!");
			}

			// convert to exclusive scan
			kernShiftRight<<<gridSize, blockSize>>>(n, dev_out, dev_in);
			checkCUDAError("kernShiftRight failed!");

			timer().endGpuTimer();

			// copy output to host
			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy odata dev_out failed!");

			hipFree(dev_in);
			hipFree(dev_out);
			checkCUDAError("hipFree failed!");
        }
    }
}
