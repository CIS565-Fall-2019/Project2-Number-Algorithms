#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <iostream>
#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
    namespace Naive {
#define blocksize 128
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		__global__ void naive_parallel_scan(unsigned long long int n, long long *odata, const long long *idata, long d) {
			unsigned long long int index = blockDim.x * blockIdx.x + threadIdx.x;
			if (index >= n)
				return;
			if (index >= d)
				odata[index] = idata[index - d] + idata[index];
			else
				odata[index] = idata[index];
		}
		__global__ void right_shift(unsigned long long int n, long long *odata, const long long *idata, int amount) {
			unsigned long long int index = blockDim.x * blockIdx.x + threadIdx.x;
			if (index >= n)
				return;
			if (index < amount)
				odata[index] = 0;
			else
				odata[index] = idata[index - amount];
		}
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(unsigned long long int n, long long *odata, const long long *idata) {
			unsigned long long int blocks = (n + blocksize - 1) / blocksize;
			// allocate data
			long long *dev_odata, *dev_odata_2;
			hipMalloc((void**)&dev_odata, n * sizeof(long long));
			checkCUDAErrorWithLine("malloc failed!");
			hipMalloc((void**)&dev_odata_2, n * sizeof(long long));
			checkCUDAErrorWithLine("malloc failed!");
			// copy data over
			hipMemcpy(dev_odata, idata, n*sizeof(long long), hipMemcpyHostToDevice);
			checkCUDAErrorWithLine("memcpy failed!");
			timer().startGpuTimer();
			unsigned long long int uppper_limit = 1 << ilog2ceil(n);
			for (long d = 1; d <= uppper_limit; d<<=1) {
				naive_parallel_scan <<<blocks, blocksize >> > (n, dev_odata_2, dev_odata, d);
				checkCUDAErrorWithLine("fn failed!");
				std::swap(dev_odata, dev_odata_2);
			}
			right_shift <<<blocks, blocksize >>> (n, dev_odata_2, dev_odata, 1);
			checkCUDAErrorWithLine("right shift failed failed!");
			hipMemcpy(odata, dev_odata_2, n*sizeof(long long), hipMemcpyDeviceToHost);
			checkCUDAErrorWithLine("memcpy back failed!");
			timer().endGpuTimer();
			hipFree(dev_odata);
			hipFree(dev_odata_2);
			// create buffer
        }
    }
}
