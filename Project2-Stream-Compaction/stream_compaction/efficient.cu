#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
		using namespace StreamCompaction::Common;

        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ 
		void kernReduction(int n, unsigned int d, int *idata)
		{
			int k = blockIdx.x * blockDim.x + threadIdx.x;
			unsigned int offset = 1 << d;

			if (k >= n || k % (offset << 1) !=0 ) return;

			idata[k + (offset << 1) - 1] += idata[k + offset - 1];
		}

		__global__
			void kernDownSweep(int n, unsigned int d, int* idata)
		{
			int k = blockIdx.x * blockDim.x + threadIdx.x;

			unsigned int offset = 1 << d;
			if (k >= n || k % (offset << 1) != 0) return;

			int tmp = idata[k + offset - 1];	// Save left child
			idata[k + offset - 1] = idata[k + (offset << 1) - 1];	// Set left child to this node's value
			idata[k + (offset << 1) - 1] += tmp;	// Set right child to old left value + this node's value
		}

		__global__ void kernSetZero(int n, int* idata)
		{
			idata[n - 1] = 0;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool useTimer) {
            // DONE
			int* d_idata;
			int maxD = ilog2ceil(n);
			int nCeil = 1 << maxD;

			hipMalloc(&d_idata, nCeil * sizeof(int));
			hipMemset(d_idata, 0, nCeil * sizeof(int));
			hipMemcpy(d_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			int threadsPerBlock = Common::THREADS_PER_BLOCK;
			int blockSize = (nCeil + threadsPerBlock - 1) / threadsPerBlock;

			if (useTimer) timer().startGpuTimer();
			// Parallel Reduction
			for (int d = 0; d < maxD; d++) {
				kernReduction << <blockSize, threadsPerBlock >> > (n, d, d_idata);
			}

			kernSetZero << <1, 1 >> > (nCeil, d_idata);
			// Down Sweep
			for (int d = maxD-1; d >= 0; d--) {
				kernDownSweep << <blockSize, threadsPerBlock >> > (n, d, d_idata);
			}
			if (useTimer) timer().endGpuTimer();

			hipMemcpy(odata, d_idata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(d_idata);

        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // DONE
			int numOfCompacted = 0;
			int* d_idata;
			int* d_odata;
			int* d_bools;
			int* d_indices;

			hipMalloc(&d_idata, n * sizeof(int));
			hipMalloc(&d_odata, n * sizeof(int));
			hipMalloc(&d_bools, n * sizeof(int));
			hipMalloc(&d_indices, n * sizeof(int));

			hipMemcpy(d_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();
			int threadsPerBlock = Common::THREADS_PER_BLOCK;
			int blockSize = (n + threadsPerBlock - 1) / threadsPerBlock;
			kernMapToBoolean<<<blockSize, threadsPerBlock>>>(n, d_bools, d_idata);

			// false: Not using the scan's timer
			scan(n, d_indices, d_bools, false);
			timer().endGpuTimer();


			int* lastIndex = d_indices + n - 1;
			hipMemcpy(&numOfCompacted, lastIndex, sizeof(int), hipMemcpyDeviceToHost);

			kernScatter << <blockSize, threadsPerBlock >> > (n, d_odata, d_idata, d_bools, d_indices);

			hipDeviceSynchronize();

			hipMemcpy(odata, d_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(d_idata);
			hipFree(d_odata);
			hipFree(d_bools);
			hipFree(d_indices);


            return (idata[n-1] > 0 ) ? ( numOfCompacted + 1 ) : numOfCompacted;
        }
    }
}
