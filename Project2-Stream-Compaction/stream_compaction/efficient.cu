#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#define blockSize 128
namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernUpsweep(int n, int d, int* idata) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int offset  =  (1 << (d + 1)); // implementing 2^d+1 incrementing 
			int k = index * offset;
			if (k >= n) {
				return;
			}

			idata[k + offset - 1] += idata[k + (1 << d) - 1];
		}

		__global__ void kernDownsweep(int n, int d, int* idata) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int offset = (1 << (d + 1)); // implementing 2^d+1 incrementing 
			int k = index * offset;
			if (k >= n) {
				return;
			}

			int t = idata[k + (1 << d) - 1];
			idata[k + (1 << d) - 1] = idata[k + offset - 1];
			idata[k + offset - 1] += t;
		}


		void printDeviceArr(int n, int* device_arr) {
			int* arr = (int*)malloc(sizeof(int)*n);
			hipMemcpy(arr, device_arr, sizeof(int) *n, hipMemcpyDeviceToHost);
			printf("\n [");
			for (int i = 0; i < n; i++) {
				printf("%d, ", arr[i]);
			}
			printf("]\n");
			free(arr);
		}
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			dim3 threadsPerBlock(blockSize);
			int npt = 1 << ilog2ceil(n);

			int * dev_idata;
			hipMalloc((void **)&dev_idata, npt * sizeof(int));
			checkCUDAError("Error: Cuda Malloc for input data on device");
			if (npt != n) {
				int *new_cpu_arr = (int*) malloc(sizeof(int) * npt);
				memset(new_cpu_arr, 0, sizeof(int) * npt);
				memcpy(new_cpu_arr, idata, sizeof(int) * n);
			    hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
				checkCUDAError("Cuda Error on non-power of two array");
			} else { 
			    hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
				checkCUDAError("Cuda Error on power of two array");
			}

            timer().startGpuTimer();
			for (int d = 0; d < ilog2ceil(npt); d++) {
				int updateThreadUtil = npt / (1 << (npt + 1));
				dim3 blocksPerGrid((updateThreadUtil + blockSize - 1) / blockSize);
				kernUpsweep << <blocksPerGrid, threadsPerBlock >> > (npt, d, dev_idata);
			}

			int z = 0;
			hipMemcpy(&dev_idata[npt - 1], &z, sizeof(int), hipMemcpyHostToDevice);
			for (int d = ilog2(npt) - 1; d >= 0; d--) {
				int updateThreadUtil = npt / (1 << (npt + 1));
				dim3 blocksPerGrid((updateThreadUtil + blockSize - 1) / blockSize);
				kernDownsweep << <blocksPerGrid, threadsPerBlock >> > (npt, d, dev_idata);
			}
            // TODO
            timer().endGpuTimer();
			hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
