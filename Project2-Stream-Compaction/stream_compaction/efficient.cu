#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		//###########################
		// MEMORY POINTERS
		//###########################
		int* kern_idata;
		int* kern_odata;
		int* kern_tdata;

		//###########################
		// FORWARD DELCARATIONS
		//###########################
		void doscan(int n, int* odata, const int* idata);

		//###########################
		// KERNEL (DEVICE) FUNCTIONS
		//###########################

		/**
		* Does the actual data movement for the upsweep
		* Organized a bit strange to make sure threads doing work are centered around the lower warps, rather than spread out
		* Not sure whether or not is more efficient
		*/
		__device__ void upsweepMove(int N, int threadnum, int currentLevel, int* idata) {
			int multiplier = 1 << currentLevel;
			unsigned long index = (multiplier * (threadnum + 1)) - 1;
			if (index >= N) return;

			multiplier = multiplier >> 1;//turns into the gap we're facing

			idata[index] = idata[index] + idata[index - multiplier];

		}//uspweepMove

		//###########################
		// KERNEL (GLOBAL) FUNCTIONS
		//###########################

		/**
		* Does the parallel-reduction upsweep
		* Needs idata to be of a power-of-two size
		* Operates in-place
		*/
		__global__ void upsweep(int N, int numLevels, int* idata) {
			int threadnum = threadIdx.x + (blockIdx.x * blockDim.x);	
			int currentLevel = 1;
			while (currentLevel <= numLevels) {
				upsweepMove(N, threadnum, currentLevel, idata);
				currentLevel++;

				__syncthreads();

			}//while
			
		}//upsweep

		//Not gonna do the fancy thread movement as before;
		//fine just wasting the work of entire warps sitting around while one thread does its stuff
		__global__ void downsweep(int N, int numLevels, int* idata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			int currentLevel = 0;
			int levelWidth;
			while (currentLevel < numLevels) {
				__syncthreads();
				levelWidth = N >> currentLevel;
				if (!((index + 1) % levelWidth)) {
					int jumpDist = levelWidth / 2;
					int temp = idata[index];
					idata[index] = idata[index] + idata[index - jumpDist];
					idata[index - jumpDist] = temp;
				}//if we're on a node this level

				currentLevel++;
			}//while

		}//downsweep

		__global__ void makeTempArray(int n, int N, int* idata, int* tdata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) return;
			if (index >= n) {
				tdata[index] = 0;
			}//if
			else {
				if (idata[index]) tdata[index] = 1;
				else tdata[index] = 0;
			}//else
		}//makeTempArray

		__global__ void scatter(int n, int* idata, int* tdata, int* odata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			if (idata[index]) {
				odata[tdata[index]] = idata[index];
			}//if we're putting this into the result
			
		}//scatter

		
		//###########################
		// CPU (MAIN) FUNCTIONS
		//###########################


		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void scan(int n, int* odata, const int* idata) {
			int numLevels = ilog2ceil(n);
			int N = 1 << numLevels;//pad out to this many elements
			int numToFake = N - n;
			int* fakeZeroes = (int*)malloc(numToFake * sizeof(int));
			for (int i = 0; i < numToFake; i++) fakeZeroes[i] = 0;

			int threadsPerBlock = (N + BLOCKSIZE - 1) / BLOCKSIZE;
			dim3 tpb = dim3(threadsPerBlock);
			dim3 bpg = dim3(BLOCKSIZE);


			hipMalloc((void**)& kern_idata, N * sizeof(int));
			checkCUDAErrorFn("hipMalloc kern_idata failed!\n", NULL, __LINE__);

			hipMemcpy(kern_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy kern_idata failed!\n", NULL, __LINE__);
			hipMemcpy(&(kern_idata[n]), fakeZeroes, numToFake * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();

			//Upsweep on kern_idata
			upsweep<<<tpb, bpg>>>(N, numLevels, kern_idata);
			checkCUDAErrorFn("upsweep failed!\n", NULL, __LINE__);

			//Downsweep on kern_idata

			//this variable does that first "set root to 0" step
			int fakeZero = 0;
			hipMemcpy(&kern_idata[N - 1], &fakeZero, sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("CudaMemcpy failed!\n", NULL, __LINE__);

			//actual downsweep
			downsweep<<<tpb, bpg>>> (N, numLevels, kern_idata);
			checkCUDAErrorFn("upsweep failed!\n", NULL, __LINE__);

			timer().endGpuTimer();

			hipMemcpy(odata, kern_idata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpy kern_idata failed!\n", NULL, __LINE__);

			hipFree(kern_idata);
			checkCUDAErrorFn("hipFree failed!\n", NULL, __LINE__);
			free(fakeZeroes);
		}//doscan

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {


			int numLevels = ilog2ceil(n);
			int N = 1 << numLevels;//pad out to this many elements
			int numToFake = N - n;

			int threadsPerBlock = (n + BLOCKSIZE - 1) / BLOCKSIZE;
			dim3 tpb = dim3(threadsPerBlock);
			dim3 bpg = dim3(BLOCKSIZE);
			int threadsPerBlockN = (N + BLOCKSIZE - 1) / BLOCKSIZE;
			dim3 tpbN = dim3(threadsPerBlock);
			dim3 bpgN = dim3(BLOCKSIZE);



			hipMalloc((void**)& kern_idata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc kern_idata failed!\n", NULL, __LINE__);
			hipMalloc((void**)& kern_tdata, N * sizeof(int));
			checkCUDAErrorFn("hipMalloc kern_odata failed!\n", NULL, __LINE__);
			hipMalloc((void**)& kern_odata, n * sizeof(int));
			checkCUDAErrorFn("hipMalloc kern_odata failed!\n", NULL, __LINE__);

			hipMemcpy(kern_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy kern_idata failed!\n", NULL, __LINE__);

			timer().startGpuTimer();

			//make our temporary binary array
			makeTempArray<<<tpbN, bpgN>>>(n, N, kern_idata, kern_tdata);
			checkCUDAErrorFn("tempArray failed!\n", NULL, __LINE__);
			hipDeviceSynchronize();

			//Scan the binary array

			//Upsweep on kern_idata
			upsweep <<<tpbN, bpgN >>> (N, numLevels, kern_tdata);
			checkCUDAErrorFn("upsweep failed!\n", NULL, __LINE__);

			//Downsweep on kern_idata

			//this variable does that first "set root to 0" step
			int fakeZero = 0;
			hipMemcpy(&kern_tdata[N - 1], &fakeZero, sizeof(int), hipMemcpyHostToDevice);
			checkCUDAErrorFn("CudaMemcpy failed!\n", NULL, __LINE__);

			//actual downsweep
			downsweep <<<tpbN, bpgN >>> (N, numLevels, kern_tdata);
			checkCUDAErrorFn("upsweep failed!\n", NULL, __LINE__);

			//scatter
			//get the ending size of the odata
			int outputSize = -1;
			hipMemcpy(&outputSize, &kern_tdata[n - 1], sizeof(int), hipMemcpyDeviceToHost);
			if (idata[n - 1]) outputSize++;//necessary because tdata holds the exclusive scan, not inclusive


			//actually scatter
			scatter<<<tpb, bpg>>>(n, kern_idata, kern_tdata, kern_odata);

			timer().endGpuTimer();

			hipMemcpy(odata, kern_odata, outputSize * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpy failed!\n", NULL, __LINE__);

            
			hipFree(kern_idata);
			checkCUDAErrorFn("hipFree failed!\n", NULL, __LINE__);
			hipFree(kern_tdata);
			checkCUDAErrorFn("hipFree failed!\n", NULL, __LINE__);
			hipFree(kern_odata);
			checkCUDAErrorFn("hipFree failed!\n", NULL, __LINE__);


			return outputSize;
        }
    }
}
