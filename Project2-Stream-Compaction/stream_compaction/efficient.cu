#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		int *dev_odata;
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

		void printArray(int n, int *a, bool abridged = false) {
			printf("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
		}

		__global__ void up_sweep(int N, int *Dev_odata, int d) {

			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			index = index * (1 << (d + 1));

			if (index > N-1) {
				return;
			}

			if (((index + (1 << (d)) - 1) < N) && ((index + (1 << (d + 1)) - 1) < N)) {

				Dev_odata[index + (1 << (d + 1)) - 1] += Dev_odata[index + (1 << (d)) - 1];
			}

			

		}


		__global__ void down_sweep(int N, int *Dev_odata, int d) {

			int index = threadIdx.x + (blockIdx.x * blockDim.x);


			index = index * (1 << (d + 1));


			if (index > N-1) {
				return;
			}


			if (((index + (1 << (d)) - 1) < N) && ((index + (1 << (d + 1)) - 1) < N)) {

				int t = Dev_odata[index + (1 << (d)) - 1];
				Dev_odata[index + (1 << (d)) - 1] = Dev_odata[index + (1 << (d + 1)) - 1];
				Dev_odata[index + (1 << (d + 1)) - 1] += t;
			}

			
		}
		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata) {


			bool timer_started = false;
			try {
				timer().startGpuTimer();
			}
			catch (const std::exception& e) {
				timer_started = true;
			}

			int blockSize = 32;
			//printArray(n, idata);
			//int new_n = n;
			n = 1 << ilog2ceil(n); // make n something that is power of 2

			hipMalloc((void**)&dev_odata, n * sizeof(int));
			hipMemcpy(dev_odata, idata, sizeof(int) * n, hipMemcpyHostToDevice);



			for (int d = 0; d <= ((ilog2ceil(n)) - 1); d++) {
				int count_thread = 1 << ((ilog2ceil(n) - d - 1));   // i need ceil(n/d) threads total
				dim3 fullBlocksPerGrid(((count_thread)+blockSize -1)/ blockSize);
				up_sweep << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, d);
			}

			hipMemset(n + dev_odata - 1, 0, sizeof(int));

			for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
				int count_thread = 1 << ((ilog2ceil(n) - d - 1));   // i need ceil(n/d) threads total
				dim3 fullBlocksPerGrid(((count_thread)+blockSize - 1) / blockSize);
				down_sweep << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, d);
			}

			
			hipMemcpy(odata, dev_odata, sizeof(int) * (n), hipMemcpyDeviceToHost);
			//hipMemcpy(dev_odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToDevice);
			//odata[0] = 0;
			//printArray(n, odata);
			if (timer_started == false) {
				timer().endGpuTimer();
			}
			hipFree(dev_odata);

		}

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact. 
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
