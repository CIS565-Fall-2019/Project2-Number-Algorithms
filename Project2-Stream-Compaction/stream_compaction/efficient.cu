#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

/*! Block size used for CUDA kernel launch*/
#define blockSize 128
namespace StreamCompaction {
	namespace Efficient {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		int nextPowerOf2(int n) {
			int p = 1;
			if (n && !(n & (n - 1))) {
				return n;
			}
			while (p < n) {
				p <<= 1;
			}
			return p;
		}

		__global__ void kernUpsweep(int n, int d, int *odata, int incr, int twod) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			//also return if index is not a multiple of the incr
			if (index >= n || (index) % incr != 0) {
				return;
			}
			//if we reached here, index+1 must be a multiple of incr (2^(d+1))
			odata[index + incr - 1] += odata[index + twod - 1];
			odata[n - 1] = 0;
		}
		__global__ void kernDownsweep(int n, int d, int *odata, int incr, int twod) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			//also return if index is not a multiple of the incr
			if (index >= n || (index) % incr != 0) {
				return;
			}
			//if we reached here, index+1 must be a multiple of incr (2^(d+1))
			int t = odata[index + twod - 1];
			odata[index + twod - 1] = odata[index + incr - 1];
			odata[index + incr - 1] += t;
		}

		__global__ void kernMapToBoolean(int n, int *mask, int *idata) {
			//dev_odata contains idata
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			if (idata[index] != 0) {
				mask[index] = 1;
			}
			else {
				mask[index] = 0;
			}
		}

		__global__ void kernScatter(int n, int *mask, int *odata, int *odata2, int *idata) {
			//odata now contains scan result
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n){
				return;
			}
			int shouldInclude = mask[index];
			if (shouldInclude) {
				int newIdx = odata2[index];
				odata[newIdx] = idata[index];
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int malloc_size = nextPowerOf2(n);
			//CUDA Malloc buffers
			int *dev_odata;
			hipMalloc((void**)&dev_odata, malloc_size * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
			int max_level = ilog2ceil(n);
			int incr = 0;
			int twod = 0;
			//Copy idata into dev_odata
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_odata failed!");

            timer().startGpuTimer();
			//Upsweep
			for (int d = 0; d < max_level; d++) {
				incr = pow(2, d + 1);
				twod = pow(2, d);
				kernUpsweep<<<fullBlocksPerGrid, blockSize >>>(malloc_size, d, dev_odata, incr, twod);
			}

			//Downsweep
			for (int d = max_level-1; d >= 0; d--) {
				incr = pow(2, d + 1);
				twod = pow(2, d);
				kernDownsweep<<<fullBlocksPerGrid, blockSize >>>(malloc_size, d, dev_odata, incr, twod);
			}
            timer().endGpuTimer();
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			//Free Memory
			hipFree(dev_odata);
        }

        void scan_notimer(int n, int malloc_size, int *dev_odata) {
			//Odata contains mask info
			dim3 fullBlocksPerGrid((malloc_size + blockSize - 1) / blockSize);
			int max_level = ilog2ceil(n);
			int incr = 0;
			int twod = 0;

			//Upsweep
			for (int d = 0; d < max_level; d++) {
				incr = pow(2, d + 1);
				twod = pow(2, d);
				kernUpsweep<<<fullBlocksPerGrid, blockSize >>>(malloc_size, d, dev_odata, incr, twod);
			}

			//Downsweep
			for (int d = max_level-1; d >= 0; d--) {
				incr = pow(2, d + 1);
				twod = pow(2, d);
				kernDownsweep<<<fullBlocksPerGrid, blockSize >>>(malloc_size, d, dev_odata, incr, twod);
			}
        }


		void printArray(int n, int *a, bool abridged = false) {
			printf("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
}

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			int malloc_size = nextPowerOf2(n);
			//CUDA Malloc buffers
			int *dev_odata;
			int *dev_odata2;
			int *dev_idata;
			int *dev_mask;
			hipMalloc((void**)&dev_odata, (malloc_size+1) * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_odata2, (malloc_size+1) * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_idata, malloc_size * sizeof(int));
			checkCUDAError("hipMalloc dev_in failed!");
			hipMalloc((void**)&dev_mask, malloc_size * sizeof(int));
			checkCUDAError("hipMalloc dev_temp failed!");

			//Memcpy idata into dev_odata for starters
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_idata failed!");
			hipMemcpy(dev_odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy dev_odata failed!");
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            timer().startGpuTimer();
			//1: Compute mask (Temporary Array)
			kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(n, dev_odata, dev_idata);

			//2: Exclusive Scan on TempArray
			hipMemcpy(dev_mask, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy dev_odata failed!");
			scan_notimer(n, malloc_size, dev_odata);

			//2.5: Get Count from dev_mask
			int tempcount[1];
			hipMemcpy(&tempcount, dev_odata + n - 1, 1 * sizeof(int), hipMemcpyDeviceToHost);
			int count = idata[n - 1] == 0 ? tempcount[0] : tempcount[0] + 1;

			//3: Scatter (dev_odata now contains scan info)
			hipMemcpy(dev_odata2, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy dev_odata failed!");
			kernScatter<<<fullBlocksPerGrid, blockSize>>>(n, dev_mask, dev_odata, dev_odata2, dev_idata);
            timer().endGpuTimer();
			hipMemcpy(odata, dev_odata, (count) * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_mask);
			hipFree(dev_odata);
			hipFree(dev_odata2);
			hipFree(dev_idata);
            return count;
        }
    }
}
