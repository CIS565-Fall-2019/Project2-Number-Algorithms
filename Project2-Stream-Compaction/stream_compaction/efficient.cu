#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void resetZeros(int n, int *a) {
			int index = (blockDim.x*blockIdx.x) + threadIdx.x;
			if (index >= n) return;
			a[index] = 0;
		}


		__global__ void upSweep(int n, int d, int *idata) {
			int index = (blockDim.x*blockIdx.x) + threadIdx.x;

			int twoPowd1 = 1 << (d + 1);
			int twoPowd = 1 << d;


			if ((index % twoPowd1 != twoPowd1-1) || index >= n) return;

			int k = index - twoPowd1 + 1;
			idata[index] += idata[k + twoPowd - 1];
		}

		__global__ void downSweep(int n, int d, int *idata) {
			int index = (blockDim.x*blockIdx.x) + threadIdx.x;

			int twoPowd1 = 1 << (d + 1);
			int twoPowd = 1 << d;


			if ((index % twoPowd1 != twoPowd1 - 1) || index >= n) return;

			int k = index - twoPowd1 + 1;
			int t = idata[k + twoPowd - 1];
			idata[k + twoPowd - 1] = idata[index];
			idata[index] += t;
		}

		void printxxx(int n, const int *a) {
			for (int i = 0; i < n; i++) {
				printf("%d ", a[i]);
			}
			printf("\n\n\n");
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			bool exception = false;
			try {
				timer().startGpuTimer();
			} catch (const std::runtime_error& ex) {
				exception = true;
			}

			int *dev_idata;

			int numThreads = 128;
			int numBlocks = (n + numThreads - 1) / numThreads;

			int d_max = ilog2ceil(n);

			int twoPowN = 1 << d_max;
			if (n != twoPowN) {

				int diff = twoPowN - n;

				hipMalloc((void **)&dev_idata, (n + diff) * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_odata1 failed!");

				resetZeros << <numBlocks, numThreads >> > (n + diff, dev_idata);

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
				n = n + diff;
			} else {
				hipMalloc((void **)&dev_idata, n * sizeof(int));
				checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

				hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			}

			for (int d = 0; d < d_max; d++) {
				upSweep<<<numBlocks, numThreads>>>(n, d, dev_idata);
			}

			// reset last element to zero
			int* zero = new int[1];
			zero[0] = 0;
			hipMemcpy(dev_idata + n - 1, zero, sizeof(int), hipMemcpyHostToDevice);

			
			for(int d = d_max-1; d >= 0; d--) {
				downSweep << <numBlocks, numThreads >> > (n, d, dev_idata);
			}


			hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
			
			hipFree(dev_idata);

			if(!exception)
				timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

			int numThreads = 128;
			int numBlocks = (n + numThreads - 1) / numThreads;

			int *dev_checkZeros, *dev_sumIndices, *dev_odata, *dev_idata;

			hipMalloc((void **) &dev_checkZeros, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_checkZeros failed!");
			hipMalloc((void **) &dev_sumIndices, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_sumIndices failed!");
			hipMalloc((void **)&dev_odata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_odata failed!");
			hipMalloc((void **)&dev_idata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_idata failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			StreamCompaction::Common::kernMapToBoolean<<<numBlocks, numThreads>>>(n, dev_checkZeros, dev_idata);
			
			int *checkZeros = new int[n];
			hipMemcpy(checkZeros, dev_checkZeros, n * sizeof(int), hipMemcpyDeviceToHost);

			//printxxx(n, checkZeros);

			int *sumIndices = new int[n];
			scan(n, sumIndices, checkZeros);

			hipMemcpy(dev_sumIndices, sumIndices , n * sizeof(int), hipMemcpyHostToDevice);

			StreamCompaction::Common::kernScatter<<<numBlocks, numThreads>>>(n, dev_odata, dev_idata, dev_checkZeros, dev_sumIndices);

			

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			

			int count = checkZeros[n - 1] == 0 ? sumIndices[n - 1] : sumIndices[n - 1] + 1;

			//delete[] checkZeros;
			//delete[] sumIndices;

			//printf("hey\n");

			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_checkZeros);
			hipFree(dev_sumIndices);

            timer().endGpuTimer();
            return count;
        }
    }
}
