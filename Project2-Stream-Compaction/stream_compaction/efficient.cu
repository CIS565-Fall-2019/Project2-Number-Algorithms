#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

# define blockSize 512

namespace StreamCompaction {
	namespace Efficient {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		int *dev_arr1;
		int *dev_arr2;
		int *dev_bools;
		int *dev_indices;
		int *dev_odata;

		__global__ void kernUpSweep(int n, int valPower2D, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n)
				return;

			if (index % (2 * valPower2D) == 0 && (index + (2 * valPower2D) - 1 < n) && (index + valPower2D - 1 < n)) {
				data[index + (2 * valPower2D) - 1] += data[index + valPower2D - 1];
			}
		}

		__global__ void kernZeroPadding(int n, int N, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n && index < N)
				data[index] = 0;
			else
				return;
		}
		__global__ void kernLastElement(int n, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index == n - 1)
				data[index] = 0;
			else
				return;
		}
		__global__ void kernDownSweep(int n, int valPower2D, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n)
				return;

			if ((index % (2 * valPower2D) == 0) && (index + (2 * valPower2D) - 1 < n) && (index + valPower2D - 1 < n)) {
				int temp = data[index + valPower2D - 1];
				data[index + valPower2D - 1] = data[index + (2 * valPower2D) - 1];
				data[index + (2 * valPower2D) - 1] += temp;
			}
		}
		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata) {

			int diff = (1 << ilog2ceil(n)) - n;
			int N = n + diff;

			hipMalloc((void**)&dev_arr1, N * sizeof(int));
			checkCUDAErrorFn("Malloc idata into arr1 failed");

			hipMemcpy(dev_arr1, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAErrorFn("Copying idata to arr1 failed");

			dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

			bool stopTimer = false;
			try {
				timer().startGpuTimer();
			}
			catch (const std::runtime_error& exception) {
				stopTimer = true;
			}


			if (diff) {
				kernZeroPadding << <fullBlocksPerGrid, blockSize >> > (n, N, dev_arr1);
			}

			for (int d = 0; d <= ilog2ceil(n) - 1; d++) {
				int valPower2D = 1 << d;
				kernUpSweep << <fullBlocksPerGrid, blockSize >> > (N, valPower2D, dev_arr1);
				checkCUDAErrorFn("Kernel Up Sweep Failed");
			}

			kernLastElement << <fullBlocksPerGrid, blockSize >> > (N, dev_arr1);

			for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
				int valPower2D = 1 << d;
				kernDownSweep << <fullBlocksPerGrid, blockSize >> > (N, valPower2D, dev_arr1);
				checkCUDAErrorFn("Kernel Down Sweep Failed");
			}


			if (!stopTimer)
				timer().endGpuTimer();

			hipMemcpy(odata, dev_arr1, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAErrorFn("Copying back to Host failed");

			hipFree(dev_arr1);


		}

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int compact(int n, int *odata, const int *idata) {

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			hipMalloc((void**)&dev_arr2, n * sizeof(int));
			checkCUDAErrorFn("Malloc idata into arr2 failed");

			hipMalloc((void**)&dev_bools, n * sizeof(int));
			checkCUDAErrorFn("Malloc idata into arr3 failed");

			hipMalloc((void**)&dev_indices, n * sizeof(int));
			checkCUDAErrorFn("Malloc idata into indices failed");

			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAErrorFn("Malloc idata into odata failed");

			hipMemcpy(dev_arr2, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAErrorFn("Copying idata to arr2 failed");

			timer().startGpuTimer();

			Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bools, dev_arr2);
			checkCUDAErrorFn("Kernel Map indicator failed");

			int *indices = new int[n];
			int *bools = new int[n];

			hipMemcpy(bools, dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAErrorFn("Copying bools to host failed");

			scan(n, indices, bools);

			hipMemcpy(dev_indices, indices, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAErrorFn("Copying indices to device failed");

			Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_arr2, dev_bools, dev_indices);
			checkCUDAErrorFn("Kernel Scatter failed");

			timer().endGpuTimer();

			int length = indices[n - 1];

			if (idata[n - 1])
				length += 1;

			//printf("Length is %d \n", length);
			hipMemcpy(odata, dev_odata, sizeof(int) * length, hipMemcpyDeviceToHost);
			checkCUDAErrorFn("Copying back to the host failed");

			hipFree(dev_arr2);
			hipFree(dev_bools);
			hipFree(dev_indices);
			hipFree(dev_odata);
			return length;
		}
	}
}